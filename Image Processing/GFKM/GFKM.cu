#include "hip/hip_runtime.h"
#include "GFKM.cuh"
#define DIM_MAX 64
#define K_MAX 32
#define M_MAX 2
#define PRN 9

__global__ void initialize_NNT_kernel
	(double* points, double* centroids, int* NNT, double* DNNT,
	 int N, int D, int K, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx >= N) return;
	int i, j;
	i = idx*M;
	int* pNNT = NNT + i;
	double* pDNNT = DNNT + i;
	double* pCentroids = centroids;
	
	double X[DIM_MAX];
	double diff, temp;
	
	j= idx*D;
	for(i=0; i<D; i++) X[i] = points[j++];

	for(i=0; i<M; i++) pDNNT[i] = DBL_MAX;

	for(i=0; i<K; i++,pCentroids+=D){
		diff = 0.0;

		for(j=0; j<D; j++){
			temp = X[j] - pCentroids[j];
			diff = diff + temp*temp;
		}
		idx = 0;

		for(; idx < M; idx++)
			if(pDNNT[idx] > diff) break;

		for(j=M-1; j>idx; j--){
			pDNNT[j] = pDNNT[j-1];
			pNNT[j] = pNNT[j-1];
		}

		if(idx < M){
			pDNNT[idx] = diff;
			pNNT[idx] = i;
		}
	}
}

__global__ void update_NNT_kernel
	(double* points, double* centroids, int* NNT, double* DNNT, double* tempU, double* D_ALG,
	 int N, int D, int K, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx >= N) return;
	int i, j;
	i = idx*M;
	int* pNNT = NNT + i;
	double* pDNNT = DNNT + i;
	i = idx*K;
	double* pTempU = tempU + i;
	double* pD_ALG = D_ALG + i;
	double* pCentroids = centroids;
	
	double X[DIM_MAX];
	double diff, temp;
	
	j= idx*D;
	for(i=0; i<D; i++) X[i] = points[j++];

	for(i=0; i<M; i++) pDNNT[i] = DBL_MAX;

	for(i=0; i<K; i++,pCentroids+=D){
		diff = 0.0;

		for(j=0; j<D; j++){
			temp = X[j] - pCentroids[j];
			diff = diff + temp*temp;
		}
		pD_ALG[i] = pTempU[i]*diff;
		idx = 0;

		for(; idx < M; idx++)
			if(pDNNT[idx] > diff) break;

		for(j=M-1; j>idx; j--){
			pDNNT[j] = pDNNT[j-1];
			pNNT[j] = pNNT[j-1];
		}

		if(idx < M){
			pDNNT[idx] = diff;
			pNNT[idx] = i;
		}
	}
}

__global__ void update_memberships_kernel
	(int* NNT, double* DNNT, double* U_ALG, double* tempU, 
	 int N, int K, int M, double fuzzifier)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx >= N) return;
	int i;
	i = idx*M;
	int* pNNT = NNT + i;
	double* pDNNT = DNNT + i;
	i = idx*K;
	double* pU = U_ALG + i;
	double* pTempU = tempU + i;
	double f = 1.0 / (fuzzifier-1.0);
	double diff, sum = 0.0;

	for(i=0; i<K; i++){
		pU[i] = 0.0;
		pTempU[i] = 0.0;
	}

	for(i=0; i<M; i++){
		idx = pNNT[i];
		diff = pDNNT[i];

		if(diff == 0.0){ 
			pU[idx] = 1.0;
			pTempU[idx] = 1.0;
			return;
		}
		diff = pow(diff, f);
		pU[idx] = diff;
		sum = sum + 1.0 / diff;
	}

	for(i=0; i<M; i++){
		idx = pNNT[i];
		diff =  1.0 / (pU[idx]*sum);
		pU[idx] = diff;
		pTempU[idx] = pow(diff, fuzzifier);
	}
}

__global__ void update_memberships_kernel
	(double* points, int* NNT, double* DNNT, double* U_ALG, double* tempU, double* tempC,
	 int N, int D, int K, int M, double fuzzifier)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx >= N) return;
	int i, j;
	i = idx*M;
	int* pNNT = NNT + i;
	double* pDNNT = DNNT + i;
	i = idx*K;
	double* pU = U_ALG + i;
	double* pTempU = tempU + i;
	i *= D;
	double* pTempC = tempC + i;
	double f = 1.0 / (fuzzifier-1.0);
	double diff, sum = 0.0;
	double X[DIM_MAX];

	idx *= D;
	for(i=0; i<D; i++) X[i] = points[idx++];

	for(i=0; i<K; i++){
		pU[i] = 0.0;
		pTempU[i] = 0.0;
	}

	for(i=0; i<M; i++){
		idx = pNNT[i];
		diff = pDNNT[i];

		if(diff == 0.0){ 
			pU[idx] = 1.0;
			pTempU[idx] = 1.0;
			return;
		}
		diff = pow(diff, f);
		pU[idx] = diff;
		sum = sum + 1.0 / diff;
	}

	for(i=0; i<M; i++, tempC+=D){
		idx = pNNT[i];
		diff =  1.0 / (pU[idx]*sum);
		pU[idx] = diff;
		pTempU[idx] = pow(diff, fuzzifier);
		diff = pTempU[idx];

		for(j=0; j<D; j++)
			pTempC[j] = X[j]*diff;
	}
}

__host__ void update_centroids
	(double* points, double* centroids, int* NNT, double* tempU,
	 int N, int D, int K, int M)
{
	int i, j, k, idx;
	int* pNNT = NNT;
	double* pTempU = tempU;
	double* pPoints = points;
	double* pCentroids;
	double* sum = new double[K]();
	memset(centroids, 0, K*D*sizeof(double));

	for(i=0; i<N; i++,pTempU+=K,pNNT+=M,pPoints+=D){
		for(j=0; j<M; j++){
			idx = pNNT[j];
			sum[idx] = sum[idx] + pTempU[idx];
			pCentroids = centroids + idx*D;

			for(k=0; k<D; k++)
				pCentroids[k] = pCentroids[k] + pTempU[idx]*pPoints[k];
		}
	}
	pCentroids = centroids;

	for(i=0; i<K; i++,pCentroids+=D)
		for(j=0; j<D; j++)
			pCentroids[j] = pCentroids[j] / sum[i];
}

__host__ double reduce_J(double* D_ALG, int N, int K)
{
	int i, NK = N*K;;
	double JK = 0.0;
	
	for(i=0; i<NK; i++)
		JK = JK + D_ALG[i];
	return JK;
}

__global__ void reduce_J_kernel(double* idata, double* odata, int n)
{
	extern __shared__ double sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x*(blockDim.x*2) + tid;
	int gridSize = blockDim.x*2*gridDim.x;
	sdata[tid] = 0;

	while(i < n){
		sdata[tid] = sdata[tid] + idata[i];

		if(i+blockDim.x < n) 
			sdata[tid] = sdata[tid] + idata[i+blockDim.x];
		i += gridSize;
	}
	__syncthreads();

	if(blockDim.x > 511){
		if(tid < 256)
			sdata[tid] = sdata[tid] + sdata[tid+256];
		__syncthreads();
	}

	if(blockDim.x > 255){
		if(tid < 128) sdata[tid] = sdata[tid] + sdata[tid+128];
		__syncthreads();
	}

	if(blockDim.x > 127){
		if(tid < 64)
			sdata[tid] = sdata[tid] + sdata[tid+64];
		__syncthreads();
	}

	if(tid < 32){
		if(blockDim.x > 63) sdata[tid] = sdata[tid] + sdata[tid + 32];
		if(blockDim.x > 31) sdata[tid] = sdata[tid] + sdata[tid + 16];
		if(blockDim.x > 15) sdata[tid] = sdata[tid] + sdata[tid + 8];
		if(blockDim.x > 7) sdata[tid] = sdata[tid] + sdata[tid + 4];
		if(blockDim.x > 3) sdata[tid] = sdata[tid] + sdata[tid + 2];
		if(blockDim.x > 1) sdata[tid] = sdata[tid] + sdata[tid + 1];
	}

	if(tid == 0) odata[blockIdx.x] = sdata[0];
}

__global__ void reduce_centroids_kernel
	(double* points, int* NNT, double* tempU, double* odata,
	 int N, int D, int K, int M)
{
	extern __shared__ double sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x*(blockDim.x*2) + tid;
	int gridSize = blockDim.x*2*gridDim.x;
	int csize = K*D;
	int usize = N*M;
	int id = tid*csize;
	int j, t;
	int cid, pid;
	double u;

	for(j=0; j<csize; j++)
		sdata[id+j] = 0.0;

	while(i < usize){
		cid = NNT[i];
		t = i/M;
		u = tempU[t*K+cid];
		cid = id + cid*D;
		pid = t*D;

		for(j=0; j<D; j++)
			sdata[cid++] += points[pid++]*u;
		t = i+blockDim.x;

		if(t < usize){
			cid = NNT[t];
			t /= M;
			u = tempU[t*K+cid];
			cid = id + cid*D;
			pid = t*D;

			for(j=0; j<D; j++)
				sdata[cid++] += points[pid++]*u;
		}
		i += gridSize;
	}
	__syncthreads();

	if(blockDim.x > 511){
		if(tid < 256){
			t = (tid+256)*csize;

			for(j=0; j<csize; j++)
				sdata[id+j] += sdata[t++];
		}
		__syncthreads();
	}

	if(blockDim.x > 255){
		if(tid < 128){
			t = (tid+128)*csize;

			for(j=0; j<csize; j++)
				sdata[id+j] += sdata[t++];
		}
		__syncthreads();
	}

	if(blockDim.x > 127){
		if(tid < 64){
			t = (tid+64)*csize;

			for(j=0; j<csize; j++)
				sdata[id+j] += sdata[t++];
		}
		__syncthreads();
	}

	if(tid < 32 && blockDim.x > 63){
		t = (tid+32)*csize;

		for(j=0; j<csize; j++)
			sdata[id+j] += sdata[t++];
	}

	if(tid < 16 && blockDim.x > 31){
		t = (tid+16)*csize;

		for(j=0; j<csize; j++)
			sdata[id+j] += sdata[t++];
	}

	if(tid < 8 && blockDim.x > 15){
		t = (tid+8)*csize;

		for(j=0; j<csize; j++)
			sdata[id+j] += sdata[t++];
	}

	if(tid < 4 && blockDim.x > 7){
		t = (tid+4)*csize;

		for(j=0; j<csize; j++)
			sdata[id+j] += sdata[t++];
	}
	if(tid < 2 && blockDim.x > 3){
		t = (tid+2)*csize;

		for(j=0; j<csize; j++)
			sdata[id+j] += sdata[t++];
	}

	if(tid < 1 && blockDim.x > 1){
		t = (tid+1)*csize;

		for(j=0; j<csize; j++)
			sdata[id+j] += sdata[t++];
	}

	if(tid == 0){
		t = blockIdx.x*csize;

		for(j=0; j<csize; j++)
			odata[t++] = sdata[j];
	}
}

__host__ void reduce
	(double* data, int num_blocks, int stride)
{
	if(stride==1){
		for(int i=1; i<num_blocks; i++)
				data[0] += data[i];
	}
	else if(stride > 1){
		double * tmp = data+stride;

		for(int i=1; i<num_blocks; i++, tmp+=stride)
			for(int j=0; j<stride; j++)
				data[j] += tmp[j];
	}
}

__host__ double* GFKM_GPU(GFKM G, int block_size, int stop_iter)
{
#pragma region Declare common variables
	int i, j;
	int DBL_SIZE = sizeof(double);
	int INT_SIZE = sizeof(int);
	
	int points_size = G.N*G.D;
	int centroids_size = G.K*G.D;
	int um_size = G.N*G.M;
	int uk_size = G.N*G.K;

	int num_blocks = (int)(ceil((double)(G.N*1.0f/block_size)));
	int nblocks1, nblocks2;
	nblocks1 = (int)(ceil((double)(uk_size*1.0/(block_size*2))));
	nblocks2 =  (int)(ceil((double)(nblocks1*1.0/(block_size*2))));

	int cbsize = 6144/centroids_size;
	int ncblocks = (int)(ceil((double)(um_size*1.0/cbsize)));
	//int smsize = 6144;

	double t;
	double total_time = 0;
	double a, newJ;

	TimingGPU tmr_GPU;
	TimingCPU tmr_CPU;
#pragma endregion

#pragma region Declare device memories
	double* d_points;
	double* d_centroids;
	double* d_DNNT;
	int* d_NNT;
	double* d_DALG;
	double* d_u;
	double* d_tempU;
	double* d_JB;
	double* d_J;
	double* d_tempC;
#pragma endregion

#pragma region Declare host pinned memories
	double* p_points;
	double* p_centroids;
	double* p_DNNT;
	int* p_NNT;
	double* p_DALG;
	double* p_u;
	double* p_tempU;
	double* p_J;
	double* p_tempC;
#pragma endregion

#pragma region Malloc device
	CudaSafeCall(hipMalloc(&d_points, points_size*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_centroids, centroids_size*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_DNNT, um_size*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_NNT, um_size*INT_SIZE));
	CudaSafeCall(hipMalloc(&d_DALG, uk_size*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_u, uk_size*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_tempU, uk_size*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_JB, nblocks1*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_J, nblocks2*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_tempC, ncblocks*centroids_size*DBL_SIZE));
	//thrust::device_ptr<double> cptr = thrust::device_pointer_cast(d_DALG);
#pragma endregion

#pragma region Malloc host
	CudaSafeCall(hipHostMalloc(&p_points, points_size*DBL_SIZE));
	CudaSafeCall(hipHostMalloc(&p_centroids, centroids_size*DBL_SIZE));
	CudaSafeCall(hipHostMalloc(&p_DNNT, um_size*DBL_SIZE));
	CudaSafeCall(hipHostMalloc(&p_NNT, um_size*INT_SIZE));
	CudaSafeCall(hipHostMalloc(&p_DALG, uk_size*DBL_SIZE));
	CudaSafeCall(hipHostMalloc(&p_u, uk_size*DBL_SIZE));
	CudaSafeCall(hipHostMalloc(&p_tempU, uk_size*DBL_SIZE));
	CudaSafeCall(hipHostMalloc(&p_tempC, ncblocks*centroids_size*DBL_SIZE));

	if(nblocks1 < block_size)
		CudaSafeCall(hipHostMalloc(&p_J, nblocks1*DBL_SIZE));
	else
		CudaSafeCall(hipHostMalloc(&p_J, nblocks2*DBL_SIZE));
#pragma endregion

#pragma region Memories copy
	memcpy(p_points, G.points, points_size*DBL_SIZE);
	memcpy(p_centroids, G.centroids, centroids_size*DBL_SIZE);
	CudaSafeCall(hipMemcpy(d_points, p_points, points_size*DBL_SIZE, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_centroids, p_centroids, centroids_size*DBL_SIZE, hipMemcpyHostToDevice));  
#pragma endregion

#pragma region Initializing NNT by GPU
	tmr_GPU.StartCounter();
	initialize_NNT_kernel<<<num_blocks, block_size>>>
		(d_points, d_centroids, d_NNT, d_DNNT, G.N, G.D, G.K, G.M);
	//CudaCheckError();
	t = tmr_GPU.GetCounter();
	total_time = total_time + t;
	cout << std::fixed << std::setprecision(PRN);
	std::cout << " Initializing NNT by GPU time: " << t << endl;  
#pragma endregion

#pragma region Main loop
	for(i=0; i<G.max_iter; i++){
		std::cout << "-------------------------------------------------------" << endl;
		std::cout << "Iteration #" << i+1 << endl;
		std::cout << "-------------------------------------------------------" << endl;

#pragma region  Updating memberships by GPU
		tmr_GPU.StartCounter();
		update_memberships_kernel<<<num_blocks, block_size>>>
			(d_NNT, d_DNNT, d_u, d_tempU, G.N, G.K, G.M, G.fuzzifier);
		//CudaCheckError();
		t = tmr_GPU.GetCounter();
		total_time = total_time + t;
		std::cout << " Updating memberships by GPU time: " << t << endl;  
#pragma endregion

#pragma region Updating centroids by GPU reduction
		/*tmr_GPU.StartCounter();
		reduce_centroids_kernel<<<ncblocks, cbsize, smsize*DBL_SIZE>>>
			(d_points, d_NNT, d_tempU, d_tempC, G.N, G.D, G.K, G.M);
		CudaSafeCall(hipMemcpyAsync(p_tempC, d_tempC, 
			ncblocks*centroids_size*DBL_SIZE, hipMemcpyDeviceToHost));
		t = tmr_GPU.GetCounter();
		total_time = total_time + t;
		std::cout << " Updating centroids by GPU time: " << t << endl;
		tmr_CPU.start();
		reduce(p_tempC, ncblocks, centroids_size);
		tmr_CPU.stop();
		t = tmr_CPU.elapsed();

		if(t > 0){
			total_time = total_time + t;
			std::cout << " Reduce centroids by CPU  time: " << t << endl;
		}*/
#pragma endregion

#pragma region Updating centroids by CPU
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(p_NNT, d_NNT, um_size*INT_SIZE, hipMemcpyDeviceToHost));
		CudaSafeCall(hipMemcpyAsync(p_tempU, d_tempU, uk_size*DBL_SIZE, hipMemcpyDeviceToHost));
		t = tmr_GPU.GetCounter();
		std::cout << " Memcpy Device NNT and TempU to Host  time: " << t << endl;
		total_time = total_time + t;
		tmr_CPU.start();
		update_centroids(p_points, p_centroids, p_NNT, p_tempU, G.N, G.D, G.K, G.M);
		tmr_CPU.stop();
		t = tmr_CPU.elapsed();
		std::cout << " Calculating new centroids by CPU time: " << t << endl;
		total_time = total_time + t;  
#pragma endregion

#pragma region Updating NNT by GPU
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(d_centroids, p_centroids, centroids_size*DBL_SIZE, hipMemcpyHostToDevice));
		//CudaSafeCall(hipMemcpyAsync(d_centroids, p_tempC, centroids_size*DBL_SIZE, hipMemcpyHostToDevice));
		update_NNT_kernel<<<num_blocks, block_size>>>
			(d_points, d_centroids, d_NNT, d_DNNT, d_tempU, d_DALG, G.N, G.D, G.K, G.M);
		//CudaCheckError();
		t = tmr_GPU.GetCounter();
		total_time = total_time + t;
		std::cout << " Updating NNT by GPU time: " << t << endl;  
#pragma endregion

#pragma region Calculating J by CPU
		/*tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(p_DALG, d_DALG, uk_size*DBL_SIZE, hipMemcpyDeviceToHost));
		t = tmr_GPU.GetCounter();
		tmr_CPU.start();
		newJ = reduce_J(p_DALG, G.N, G.K);*/  
#pragma endregion

#pragma region Calculating J by thrust::reduce
		// 
		/*tmr_CPU.start();
		newJ = thrust::reduce(cptr, cptr + uk_size);*/  
#pragma endregion

#pragma region Calculating J by GPU reduction
		// Reduction J #1
		tmr_GPU.StartCounter();
		//reduce_J_kernel<<<nblocks1, block_size, block_size*DBL_SIZE>>>(d_DALG, d_JB, uk_size);
		reduce_J_kernel1<256><<<nblocks1, block_size, block_size*DBL_SIZE>>>(d_DALG, d_JB, uk_size);
		//CudaCheckError();

		if(nblocks1 < block_size){
			CudaSafeCall(hipMemcpyAsync(p_J, d_JB, nblocks1*DBL_SIZE, hipMemcpyDeviceToHost));
			t = tmr_GPU.GetCounter();
			total_time = total_time + t;
			std::cout << " Calculating J by GPU reduction time: " << t << endl;
			// Reduction J blocksums by CPU
			newJ = 0.0;
			tmr_CPU.start();

			for(j=0; j<nblocks1; j++)
				newJ = newJ + p_J[j];
		}  
		else{
			// Reduction J #2
			//reduce_J_kernel<<<nblocks2, block_size,  block_size*DBL_SIZE>>>(d_JB, d_J, nblocks1);
			reduce_J_kernel1<256><<<nblocks2, block_size,  block_size*DBL_SIZE>>>(d_JB, d_J, nblocks1);
			//CudaCheckError();
			CudaSafeCall(hipMemcpyAsync(p_J, d_J, nblocks2*DBL_SIZE, hipMemcpyDeviceToHost));
			t = tmr_GPU.GetCounter();
			total_time = total_time + t;
			std::cout << " Calculate J by GPU reduction time: " << t << endl;
			// Reduction J blocksums by CPU
			newJ = 0.0;
			tmr_CPU.start();

			for(j=0; j<nblocks2; j++)
				newJ = newJ + p_J[j];
		}  
#pragma endregion

#pragma region Getting time and checking stop-condition
		tmr_CPU.stop();
		t = tmr_CPU.elapsed();

		if(t > 0){
			total_time = total_time + t;
			std::cout << " Reduce J by CPU  time: " << t << endl;
		}
		a = fabs(newJ - G.J);
		std::cout << " Diff = " << a << endl;

		if((a < G.epsilon && (stop_iter == INT_MAX || i==stop_iter)) || i==stop_iter)
			break;
		G.J = newJ;  
#pragma endregion
	}
	if(i == G.max_iter) i--;
#pragma endregion

#pragma region Copying device back to host
		tmr_GPU.StartCounter();
	CudaSafeCall(hipMemcpyAsync(p_centroids, d_centroids, centroids_size*DBL_SIZE, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpyAsync(p_NNT, d_NNT, um_size*INT_SIZE, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpyAsync(p_DNNT, d_DNNT, um_size*DBL_SIZE, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpyAsync(p_u, d_u, uk_size*DBL_SIZE, hipMemcpyDeviceToHost));
	t = tmr_GPU.GetCounter();
	total_time += t;
	std::cout << "Device to host time: " << t << endl;
#pragma endregion

#pragma region Writing results to files
	G.write<double>(p_centroids, G.K, G.D, G.path + "centroids.GPU.txt");
	G.write<int>(p_NNT, G.N, G.M, G.path + "NNT.GPU.txt");
	G.write<double>(p_DNNT, G.N, G.M, G.path + "DNNT.GPU.txt");
	G.write<double>(p_u, G.N, G.K, G.path + "u.GPU.txt");
	G.write<double>(p_tempU, G.N, G.K, G.path + "tempU.GPU.txt");  
#pragma endregion

#pragma region Cuda free device memories
	hipFree(d_points);
	hipFree(d_centroids);
	hipFree(d_NNT);
	hipFree(d_DNNT);
	hipFree(d_DALG);
	hipFree(d_u);
	hipFree(d_tempU);
	hipFree(d_J);
	hipFree(d_JB);  
#pragma endregion

#pragma region Cuda free host pinned memories
	hipHostFree(p_points);
	hipHostFree(p_centroids);
	hipHostFree(p_NNT);
	hipHostFree(p_DNNT);
	hipHostFree(p_DALG);
	hipHostFree(p_u);
	hipHostFree(p_tempU);
	hipHostFree(p_J);  
#pragma endregion

#pragma region Returning last iteration index and total running time
	double * rs = new double[2];
	rs[0] = total_time;
	rs[1] = (double)i;  
#pragma endregion

	hipDeviceReset();
	return rs;
}