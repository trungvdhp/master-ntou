#include "hip/hip_runtime.h"
#include "GFKM.cuh"
#define DIM_MAX 36
#define PRN 9
#define BLOCK_MAX 1024

__global__ void initialize_NNT_kernel
	(double* points, double* centroids, int* NNT, double* DNNT,
	 int N, int D, int K, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx >= N) return;
	int i, j;
	i = idx*M;
	int* pNNT = NNT + i;
	double* pDNNT = DNNT + i;
	double* pCentroids = centroids;
	
	double X[DIM_MAX];
	double diff, temp;
	
	j= idx*D;
	for(i=0; i<D; i++) X[i] = points[j++];

	for(i=0; i<M; i++) pDNNT[i] = DBL_MAX;

	for(i=0; i<K; i++,pCentroids+=D){
		diff = 0.0;

		for(j=0; j<D; j++){
			temp = X[j] - pCentroids[j];
			diff = diff + temp*temp;
		}
		idx = 0;

		for(; idx < M; idx++)
			if(pDNNT[idx] > diff) break;

		for(j=M-1; j>idx; j--){
			pDNNT[j] = pDNNT[j-1];
			pNNT[j] = pNNT[j-1];
		}

		if(idx < M){
			pDNNT[idx] = diff;
			pNNT[idx] = i;
		}
	}
}

__global__ void update_memberships_kernel
	(int* NNT, double* DNNT, double* U_ALG, double* tempU, 
	 int N, int K, int M, double fuzzifier)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx >= N) return;
	int i;
	i = idx*M;
	int* pNNT = NNT + i;
	double* pDNNT = DNNT + i;
	i = idx*K;
	double* pU = U_ALG + i;
	double* pTempU = tempU + i;
	double f = 1.0 / (fuzzifier-1.0);
	double diff, sum = 0.0;

	for(i=0; i<K; i++){
		pU[i] = 0.0;
		pTempU[i] = 0.0;
	}

	for(i=0; i<M; i++){
		idx = pNNT[i];
		diff = pDNNT[i];

		if(diff == 0.0){ 
			pU[idx] = 1.0;
			pTempU[idx] = 1.0;
			return;
		}
		diff = pow(diff, f);
		pU[idx] = diff;
		sum = sum + 1.0 / diff;
	}

	for(i=0; i<M; i++){
		idx = pNNT[i];
		diff =  1.0 / (pU[idx]*sum);
		pU[idx] = diff;
		pTempU[idx] = pow(diff, fuzzifier);
	}
}

__host__ void update_centroids
	(double* points, double* centroids, int* NNT, double* tempU,
	 int N, int D, int K, int M)
{
	int i, j, k, idx;
	int* pNNT = NNT;
	double* pTempU = tempU;
	double* pPoints = points;
	double* pCentroids;
	double* sum = new double[K]();
	memset(centroids, 0, K*D*sizeof(double));

	for(i=0; i<N; i++,pTempU+=K,pNNT+=M,pPoints+=D){
		for(j=0; j<M; j++){
			idx = pNNT[j];
			sum[idx] = sum[idx] + pTempU[idx];
			pCentroids = centroids + idx*D;

			for(k=0; k<D; k++)
				pCentroids[k] = pCentroids[k] + pTempU[idx]*pPoints[k];
		}
	}
	pCentroids = centroids;

	for(i=0; i<K; i++,pCentroids+=D)
		for(j=0; j<D; j++)
			pCentroids[j] = pCentroids[j] / sum[i];
}

__global__ void histogram_kernel(int* NNT, int* histo, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < size)
		atomicAdd(&(histo[NNT[i]+1]), 1);
}

__global__ void scan_kernel(int* histo, int K)
{
	for(int i=1; i<K; i++)
		histo[i] += histo[i-1];
}

__global__ void counting_sort_kernel(int* NNT, int* histo, int* odata, int size, int M)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < size){
		int idx = atomicAdd(&(histo[NNT[i]]), 1);
		odata[idx] = i/M;
	}
}

__global__ void reduce_memberships_kernel
	(int* NNT, double* tempU, double* odata, int N, int K, int cid)
{
	extern __shared__ double sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x*blockDim.x + tid;
	int gridSize = blockDim.x*gridDim.x;
	double temp = 0.0;

	while(i < N){
		temp = temp + tempU[NNT[i]*K+cid];
		i += gridSize;
	}
	sdata[tid] = temp;
	__syncthreads();

	//if(blockDim.x > 255){
		if(tid < 128) sdata[tid] = sdata[tid] + sdata[tid+128];
		__syncthreads();
	//}

	//if(blockDim.x > 127){
		if(tid < 64)
			sdata[tid] = sdata[tid] + sdata[tid+64];
		__syncthreads();
	//}

	if(tid < 32) sdata[tid] = sdata[tid] + sdata[tid + 32];
	if(tid < 16) sdata[tid] = sdata[tid] + sdata[tid + 16];
	if(tid < 8) sdata[tid] = sdata[tid] + sdata[tid + 8];
	if(tid < 4) sdata[tid] = sdata[tid] + sdata[tid + 4];
	if(tid < 2) sdata[tid] = sdata[tid] + sdata[tid + 2];

	if(tid == 0) odata[blockIdx.x] = sdata[0] + sdata[1];
}

__global__ void reduce_centroids_kernel
	(double* points, int* NNT, double* tempU, double* odata,
	 int N, int D, int K, int pid, int cid)
{
	extern __shared__ double sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x*blockDim.x + tid;
	int gridSize = blockDim.x*gridDim.x;
	int t;
	double temp = 0.0;
	
	while(i < N){
		t = NNT[i];
		temp = temp + points[t*D + pid]*tempU[t*K+cid];
		i += gridSize;
	}
	sdata[tid] = temp;
	__syncthreads();

	//if(blockDim.x > 255){
		if(tid < 128) sdata[tid] = sdata[tid] + sdata[tid+128];
		__syncthreads();
	//}

	//if(blockDim.x > 127){
		if(tid < 64) sdata[tid] = sdata[tid] + sdata[tid+64];
		__syncthreads();
	//}

	if(tid < 32) sdata[tid] = sdata[tid] + sdata[tid + 32];
	if(tid < 16) sdata[tid] = sdata[tid] + sdata[tid + 16];
	if(tid < 8) sdata[tid] = sdata[tid] + sdata[tid + 8];
	if(tid < 4) sdata[tid] = sdata[tid] + sdata[tid + 4];
	if(tid < 2) sdata[tid] = sdata[tid] + sdata[tid + 2];

	if(tid == 0) odata[blockIdx.x] = sdata[0] + sdata[1];
}

__global__ void reduce_centroids_kernel
	(double* centroids, double* memberships, double* tempC, int* histo,  int D, int K)
{
	int tid = threadIdx.x;

	if(tid >= K) return;
	double* C = centroids + tid*D;
	int size = histo[tid];
	double a = 0.0;

	if(tid>0) tid = histo[tid-1];
	int i;
	double* tempU = memberships + tid;
	double* temp = tempC + tid*D;
	size -= tid;
	
	for(i=0; i<size; i++)
		a = a + tempU[i];

	for(i=0; i<D; i++,temp+=size){
		C[i] = 0.0;

		for(tid=0; tid<size; tid++)
			C[i] = C[i] + temp[tid];
		C[i] = C[i]/a;
	}
}

inline __host__ void reduce_centroids
	(double* centroids, double* u, double* tempC, int* histo, int D, int K)
{
	int i, j, t, x=0, y=0, cid=0;
	double a;

	for(i=0; i<K; i++){
		a = 0.0;

		for(j=0; j<histo[i]; j++,x++)
			a = a + u[x];

		for(j=0; j<D; j++,cid++){
			centroids[cid] = 0.0;

			for(t=0; t<histo[i]; t++,y++)
				centroids[cid] = centroids[cid] + tempC[y];
			centroids[cid] = centroids[cid]/a;
		}
	}
}

__global__ void update_NNT_kernel
	(double* points, double* centroids, int* NNT, double* DNNT, double* tempU, double* D_ALG,
	 int N, int D, int K, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx >= N) return;
	int i, j;
	i = idx*M;
	int* pNNT = NNT + i;
	double* pDNNT = DNNT + i;
	i = idx*K;
	double* pTempU = tempU + i;
	double* pD_ALG = D_ALG + i;
	double* pCentroids = centroids;
	
	double X[DIM_MAX];
	double diff, temp;
	
	j= idx*D;
	for(i=0; i<D; i++) X[i] = points[j++];

	for(i=0; i<M; i++) pDNNT[i] = DBL_MAX;

	for(i=0; i<K; i++,pCentroids+=D){
		diff = 0.0;

		for(j=0; j<D; j++){
			temp = X[j] - pCentroids[j];
			diff = diff + temp*temp;
		}
		pD_ALG[i] = pTempU[i]*diff;
		idx = 0;

		for(; idx < M; idx++)
			if(pDNNT[idx] > diff) break;

		for(j=M-1; j>idx; j--){
			pDNNT[j] = pDNNT[j-1];
			pNNT[j] = pNNT[j-1];
		}

		if(idx < M){
			pDNNT[idx] = diff;
			pNNT[idx] = i;
		}
	}
}

__host__ double reduce_J(double* D_ALG, int N, int K)
{
	int i, NK = N*K;;
	double JK = 0.0;
	
	for(i=0; i<NK; i++)
		JK = JK + D_ALG[i];
	return JK;
}

__global__ void reduce_J_kernel(double* idata, double* odata, int n)
{
	extern __shared__ double sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x*blockDim.x + tid;
	int gridSize = blockDim.x*gridDim.x;
	double temp = 0.0;

	while(i < n){
		temp = temp + idata[i];
		i += gridSize;
	}
	sdata[tid] = temp;
	__syncthreads();

	//if(blockDim.x > 511){
	//	if(tid < 256)
	//		sdata[tid] = sdata[tid] + sdata[tid+256];
	//	__syncthreads();
	//}

	if(blockDim.x > 255){
		if(tid < 128) sdata[tid] = sdata[tid] + sdata[tid+128];
		__syncthreads();
	}

	if(blockDim.x > 127){
		if(tid < 64) sdata[tid] = sdata[tid] + sdata[tid+64];
		__syncthreads();
	}

	if(tid < 32) sdata[tid] = sdata[tid] + sdata[tid + 32];
	if(tid < 16) sdata[tid] = sdata[tid] + sdata[tid + 16];
	if(tid < 8) sdata[tid] = sdata[tid] + sdata[tid + 8];
	if(tid < 4) sdata[tid] = sdata[tid] + sdata[tid + 4];
	if(tid < 2) sdata[tid] = sdata[tid] + sdata[tid + 2];

	if(tid == 0) odata[blockIdx.x] = sdata[0] + sdata[1];
}

inline __host__ int roundup(int x, int y)
{
	return 1 + (x-1)/y;
}

__host__ double* GFKM_GPU(GFKM G, int block_size, int stop_iter)
{
#pragma region Declare common variables
	int i, j, x, y, z, q;
	int DBL_SIZE = sizeof(double);
	int INT_SIZE = sizeof(int);
	
	int points_size = G.N*G.D;
	int centroids_size = G.K*G.D;
	int centroids_dbl_size = centroids_size*DBL_SIZE;
	int um_size = G.N*G.M;
	int uk_size = G.N*G.K;
	int sm_size = block_size*DBL_SIZE;
	int block_dsize = block_size<<2;
	int histo_size = (G.K+1)*INT_SIZE;

	int num_blocks = roundup(G.N, block_size);
	int num_Jblocks1 = roundup(uk_size, block_dsize);
	int num_Jblocks2 =  roundup(num_Jblocks1,block_dsize);
	int num_cblocks = roundup(um_size, block_dsize) + G.K;
	int num_hblocks = roundup(um_size, block_size);

	int Jblock1_size = num_Jblocks1 * DBL_SIZE;
	int Jblock2_size = num_Jblocks2 * DBL_SIZE;

	double a, newJ;
	double t, tt;
	double total_time = 0.0;
	
	TimingGPU tmr_GPU;
	TimingCPU tmr_CPU;
#pragma endregion

#pragma region Declare device memories
	double* d_points;
	double* d_centroids;
	double* d_DNNT;
	int* d_NNT;
	int* d_histo;
	int* d_sp;
	double* d_DALG;
	double* d_u;
	double* d_tempU;
	double* d_JB;
	double* d_J;
	double* d_tempC;
	double* d_m;
#pragma endregion

#pragma region Declare host pinned memories
	double* p_points;
	double* p_centroids;
	double* p_DNNT;
	int* p_NNT;
	int* p_histo;
	double* p_DALG;
	double* p_u;
	double* p_tempU;
	double* p_J;
	double* p_tempC;
	double* p_m;
#pragma endregion

#pragma region Malloc device
	CudaSafeCall(hipMalloc(&d_points, points_size*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_centroids, centroids_size*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_DNNT, um_size*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_NNT, um_size*INT_SIZE));
	CudaSafeCall(hipMalloc(&d_sp, um_size*INT_SIZE));
	CudaSafeCall(hipMalloc(&d_histo, (G.K+1)*INT_SIZE));
	CudaSafeCall(hipMalloc(&d_DALG, uk_size*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_u, uk_size*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_tempU, uk_size*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_JB, num_Jblocks1*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_J, num_Jblocks2*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_tempC, num_cblocks*centroids_size*DBL_SIZE));
	CudaSafeCall(hipMalloc(&d_m, num_cblocks*DBL_SIZE));
	//thrust::device_ptr<double> cptr = thrust::device_pointer_cast(d_DALG);
	//thrust::device_ptr<int> cptr = thrust::device_pointer_cast(d_histo);
#pragma endregion

#pragma region Malloc host
	CudaSafeCall(hipHostMalloc(&p_points, points_size*DBL_SIZE));
	CudaSafeCall(hipHostMalloc(&p_centroids, centroids_size*DBL_SIZE));
	CudaSafeCall(hipHostMalloc(&p_DNNT, um_size*DBL_SIZE));
	CudaSafeCall(hipHostMalloc(&p_NNT, um_size*INT_SIZE));
	CudaSafeCall(hipHostMalloc(&p_histo, (G.K+1)*INT_SIZE));
	CudaSafeCall(hipHostMalloc(&p_DALG, uk_size*DBL_SIZE));
	CudaSafeCall(hipHostMalloc(&p_u, uk_size*DBL_SIZE));
	CudaSafeCall(hipHostMalloc(&p_tempU, uk_size*DBL_SIZE));
	CudaSafeCall(hipHostMalloc(&p_tempC, num_cblocks*centroids_size*DBL_SIZE));
	CudaSafeCall(hipHostMalloc(&p_m, num_cblocks*DBL_SIZE));

	if(num_Jblocks1 < BLOCK_MAX)
		CudaSafeCall(hipHostMalloc(&p_J, num_Jblocks1*DBL_SIZE));
	else
		CudaSafeCall(hipHostMalloc(&p_J, num_Jblocks2*DBL_SIZE));
#pragma endregion

#pragma region Memories copy
	memcpy(p_points, G.points, points_size*DBL_SIZE);
	memcpy(p_centroids, G.centroids, centroids_size*DBL_SIZE);
	CudaSafeCall(hipMemcpy(d_points, p_points, points_size*DBL_SIZE, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_centroids, p_centroids, centroids_size*DBL_SIZE, hipMemcpyHostToDevice));  
#pragma endregion

#pragma region Cuda stream create
	hipStream_t *streams = new hipStream_t[G.D+1];
	for(i = 0; i<=G.D; i++)
		hipStreamCreate(&streams[i]);  
#pragma endregion

#pragma region Initializing NNT by GPU
	tmr_GPU.StartCounter();
	initialize_NNT_kernel<<<num_blocks, block_size>>>
		(d_points, d_centroids, d_NNT, d_DNNT, G.N, G.D, G.K, G.M);
	//CudaCheckError();
	t = tmr_GPU.GetCounter();
	total_time = total_time + t;
	cout << std::fixed << std::setprecision(PRN);
	std::cout << " Initializing NNT by GPU: " << t << endl;  
#pragma endregion

#pragma region Main loop
	for(i=0; i<G.max_iter; i++){
		std::cout << "-------------------------------------------------------" << endl;
		std::cout << "Iteration #" << i+1 << endl;
		std::cout << "-------------------------------------------------------" << endl;

#pragma region  Updating memberships by GPU
		tmr_GPU.StartCounter();
		update_memberships_kernel<<<num_blocks, block_size>>>
			(d_NNT, d_DNNT, d_u, d_tempU, G.N, G.K, G.M, G.fuzzifier);
		//CudaCheckError();
		t = tmr_GPU.GetCounter();
		total_time = total_time + t;
		std::cout << " Updating memberships by GPU: " << t << endl;  
#pragma endregion

#pragma region Updating centroids
		std::cout << " Updating centroids:"<< endl;
		tt = 0.0;
#pragma region Counting sort by GPU
		tmr_GPU.StartCounter();
		hipMemset(d_histo, 0, histo_size);
		// Calculate the histogram of cluster index frequencies from NNT
		histogram_kernel<<<num_hblocks, block_size>>>(d_NNT, d_histo, um_size);
		CudaSafeCall(hipMemcpyAsync(p_histo, d_histo, histo_size, hipMemcpyDeviceToHost));
		// Calculate the starting index for each cluster
		scan_kernel<<<1, 1>>>(d_histo, G.K);
		// Copy corresponding point indices to output array, 
		// preserving order of inputs (array NNT) with equal cluster indices
		counting_sort_kernel<<<num_hblocks, block_size>>>(d_NNT, d_histo, d_sp, um_size, G.M);
		t = tmr_GPU.GetCounter();
		tt = tt + t;
		std::cout << "  + Counting sort by GPU: " << t << endl;
#pragma endregion

#pragma region Reducing centroids by GPU
		y = 0; z = 0; q = 0;
		tmr_GPU.StartCounter();
		for(j=0; j<G.K; j++){
			p_histo[j] = roundup(p_histo[j+1], block_dsize);
			reduce_memberships_kernel<<<p_histo[j], block_size, sm_size, streams[0]>>>
				(d_sp+q, d_tempU, d_m+y, p_histo[j+1], G.K, j);

			for(x=0; x<G.D; x++){
				reduce_centroids_kernel<<<p_histo[j], block_size, sm_size, streams[x+1]>>>
					(d_points, d_sp+q, d_tempU, d_tempC+z, p_histo[j+1], G.D, G.K, x, j);
				z += p_histo[j];
			}
			y += p_histo[j];
			//p_histo[j] = y;
			q += p_histo[j+1];
		}
		CudaSafeCall(hipMemcpyAsync(p_m, d_m, y*DBL_SIZE, hipMemcpyDeviceToHost));
		CudaSafeCall(hipMemcpyAsync(p_tempC, d_tempC, z*DBL_SIZE, hipMemcpyDeviceToHost));
		t = tmr_GPU.GetCounter();
		tt = tt + t;
		std::cout << "  + Reducing centroids by GPU: " << t << endl;
#pragma endregion

#pragma region Reducing centroids by CPU
		tmr_CPU.start();
		reduce_centroids(p_centroids, p_m, p_tempC, p_histo, G.D, G.K);
		tmr_CPU.stop();
		t = tmr_CPU.elapsed();
		tt = tt + t;
		std::cout << "  + Reducing centroids by CPU: " << t << endl;
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(d_centroids, p_centroids, centroids_dbl_size, hipMemcpyHostToDevice));
		t = tmr_GPU.GetCounter();
		tt = tt + t;
		total_time = total_time + tt;
		std::cout << "  + Host to device: " << t << endl;
		std::cout << " Total updating centroids time: " << tt << endl;
#pragma endregion

#pragma endregion

#pragma region Updating NNT by GPU
		tmr_GPU.StartCounter();
		update_NNT_kernel<<<num_blocks, block_size>>>
			(d_points, d_centroids, d_NNT, d_DNNT, d_tempU, d_DALG, G.N, G.D, G.K, G.M);
		//CudaCheckError();
		t = tmr_GPU.GetCounter();
		total_time = total_time + t;
		std::cout << " Updating NNT by GPU: " << t << endl;  
#pragma endregion

#pragma region Calculating distortion value J
		std::cout << " Calculating distortion value J:" << endl;
#pragma region Calculating J by CPU
		/*tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(p_DALG, d_DALG, uk_size*DBL_SIZE, hipMemcpyDeviceToHost));
		t = tmr_GPU.GetCounter();
		tmr_CPU.start();
		newJ = reduce_J(p_DALG, G.N, G.K);*/  
#pragma endregion

#pragma region Calculating J by GPU reduction
		tt = 0.0;
		// Reduction J #1
		tmr_GPU.StartCounter();
		reduce_J_kernel<<<num_Jblocks1, block_size, sm_size>>>(d_DALG, d_JB, uk_size);
		//CudaCheckError();

		if(num_Jblocks1 < BLOCK_MAX){
			CudaSafeCall(hipMemcpyAsync(p_J, d_JB, Jblock1_size, hipMemcpyDeviceToHost));
			t = tmr_GPU.GetCounter();
			tt = tt + t;
			std::cout << "  + Reducing J by GPU: " << t << endl;
			// Reduction J blocksums by CPU
			tmr_CPU.start();
			newJ = 0.0;

			for(j=0; j<num_Jblocks1; j++)
				newJ = newJ + p_J[j];
		}  
		else{
			// Reduction J #2
			reduce_J_kernel<<<num_Jblocks2, block_size,  sm_size>>>(d_JB, d_J, num_Jblocks1);
			//CudaCheckError();
			CudaSafeCall(hipMemcpyAsync(p_J, d_J, Jblock2_size, hipMemcpyDeviceToHost));
			t = tmr_GPU.GetCounter();
			tt = tt + t;
			std::cout << "  + Reducing J by GPU: " << t << endl;
			// Reduction J blocksums by CPU
			tmr_CPU.start();
			newJ = 0.0;

			for(j=0; j<num_Jblocks2; j++)
				newJ = newJ + p_J[j];
		}
#pragma endregion
#pragma endregion

#pragma region Getting and checking stop-condition
		tmr_CPU.stop();
		t = tmr_CPU.elapsed();
		if(t < 0) t = 0;
		tt = tt + t;
		std::cout << "  + Reducing J by CPU : " << t << endl;
		total_time = total_time + tt;
		std::cout << " Total calculating J time: " << tt << endl;
		a = fabs(newJ - G.J);
		std::cout << " Difference: " << a << endl;

		if((a < G.epsilon && (stop_iter == INT_MAX || i==stop_iter)) || i==stop_iter)
			break;
		G.J = newJ;
#pragma endregion
	}
	if(i == G.max_iter) i--;
#pragma endregion

#pragma region Copying device back to host
	tmr_GPU.StartCounter();
	CudaSafeCall(hipMemcpyAsync(p_NNT, d_NNT, um_size*INT_SIZE, hipMemcpyDeviceToHost));
	//CudaSafeCall(hipMemcpyAsync(p_DNNT, d_DNNT, um_size*DBL_SIZE, hipMemcpyDeviceToHost));
	//CudaSafeCall(hipMemcpyAsync(p_u, d_u, uk_size*DBL_SIZE, hipMemcpyDeviceToHost));
	//CudaSafeCall(hipMemcpyAsync(p_tempU, d_tempU, uk_size*DBL_SIZE, hipMemcpyDeviceToHost));
	t = tmr_GPU.GetCounter();
	total_time = total_time + t;
	std::cout << " Device to host: " << t << endl;
#pragma endregion

#pragma region Writing results to files
	G.write<double>(p_centroids, G.K, G.D, G.path + "centroids.GPU.txt");
	G.write<int>(p_NNT, G.N, G.M, G.path + "NNT.GPU.txt");
	//G.write<double>(p_DNNT, G.N, G.M, G.path + "DNNT.GPU.txt");
	//G.write<double>(p_u, G.N, G.K, G.path + "u.GPU.txt");
	//G.write<double>(p_tempU, G.N, G.K, G.path + "tempU.GPU.txt");  
#pragma endregion

#pragma region Cuda free device memories
	hipFree(d_points);
	hipFree(d_centroids);
	hipFree(d_tempC);
	hipFree(d_NNT);
	hipFree(d_m);
	hipFree(d_histo);
	hipFree(d_DNNT);
	hipFree(d_DALG);
	hipFree(d_u);
	hipFree(d_tempU);
	hipFree(d_J);
	hipFree(d_JB);
	hipFree(d_sp);
#pragma endregion

#pragma region Cuda free host pinned memories
	hipHostFree(p_points);
	hipHostFree(p_centroids);
	hipHostFree(p_tempC);
	hipHostFree(p_NNT);
	hipHostFree(p_histo);
	hipHostFree(p_DNNT);
	hipHostFree(p_DALG);
	hipHostFree(p_u);
	hipHostFree(p_m);
	hipHostFree(p_tempU);
	hipHostFree(p_J);
#pragma endregion

#pragma region Get total time and last iteration index
	double *rs = new double[2];
	rs[0] = total_time;
	rs[1] = (double)i;
#pragma endregion

#pragma region CudaStreamDestroy and DeviceReset
	for(i = 0; i<=G.D; i++)
		hipStreamDestroy(streams[i]);
	hipDeviceReset();
#pragma endregion
	return rs;  
}