#include "GFKM.cuh"
#include "GFKM.h"
#include "TimingCPU.h"
#include <conio.h>

int main(int argc, char* argv[])
{
	string path = "D:\\Projects\\master-ntou\\ImageProcessing\\GFKM\\Data\\LenaPeppersBaboon\\";//sat\\";
	string fname = "LenaPeppersBaboon.txt";//"sat.data";
	int M = 2;
	int max_iter = 1000;
	double epsilon = numeric_limits<double>::epsilon();
	int mode = 1;
	int stop_iter = INT_MAX;

	if (argc > 2){
		path = string(argv[1]);
		fname = string(argv[2]);
		cout << path << endl;
		if (argc > 3){
			M = atoi(argv[3]);

			if (argc > 4){
				max_iter = atoi(argv[4]);

				if (argc > 5){
					epsilon = atof(argv[5]);
					if (argc > 6){
						mode = atoi(argv[6]);
						if (argc > 7){
							stop_iter = atoi(argv[7]) - 1;
						}
					}
				}
			}
		}
	}
	printf("mode = %d\n", mode);
	GFKM G(path, fname);
	G.epsilon = epsilon;
	G.max_iter = max_iter;
	G.M = M;
	G.initialize_centroids();
	cout << "-------------------------------------------------------" << endl;
	cout << "GPU running" << endl;
	cout << "-------------------------------------------------------" << endl;
	double * gpu_rs = GFKM_GPU(G, 256, stop_iter, mode);
	G.J = 0.0;
	cout << "-------------------------------------------------------" << endl;
	cout << "CPU running" << endl;
	cout << "-------------------------------------------------------" << endl;
	double * cpu_rs = G.run((int)gpu_rs[1]);
	cout << "-------------------------------------------------------" << endl;
	cout << "Total CPU time: " << cpu_rs[0] << endl;
	cout << "Total GPU time: " << gpu_rs[0] << endl;
	cout << "-------------------------------------------------------" << endl;
	cout << "Total CPU time / Total GPU time: " << cpu_rs[0]/gpu_rs[0] << endl;
	cout << "-------------------------------------------------------" << endl;
	delete [] gpu_rs;
	delete [] cpu_rs;
	//getch();
    return 0;
}