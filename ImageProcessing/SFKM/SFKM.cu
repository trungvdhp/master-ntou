#include "hip/hip_runtime.h"
#include "SFKM.cuh"
#include "Util.h"
#define DIM_MAX 16
#define NSTREAM 5
#define TILE_DIM 16

inline __host__ int roundup(int x, int y)
{
	return 1 + (x-1)/y;
}

__global__ void update_memberships_kernel(
	double * points, double * centroids, double * memberships, 
	int N, int D, int K, double fuzzifier)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= N) return;
	int i, j;
	double * pMemberships = memberships + idx*K;
	double * pCentroids = centroids;
	double X[DIM_MAX];
	double f = 1. / (fuzzifier - 1.);
	double diff, temp, sum = 0.;

	for (i = 0, j = idx*D; i < D; ++i, ++j) X[i] = points[j];

	for (i = 0; i < K; ++i) pMemberships[i] = 0.;

	for (i = 0; i < K; ++i, pCentroids += D){
		diff = 0.;

		for (j = 0; j < D; ++j){
			temp = X[j] - pCentroids[j];
			diff = diff + temp*temp;
		}

		if (diff == 0.){ 
			pMemberships[i] = 1.;
			return;
		}
		diff = pow(diff, f);
		pMemberships[i] = diff;
		sum = sum + 1. / diff;
	}

	for (i = 0; i < K; ++i){
		pMemberships[i] = pow(pMemberships[i]*sum, -fuzzifier);
	}
}

__global__ void MatMul(double * A, double * B, double * C, int ARows, int ACols, int BCols) 
{
    double CValue = 0;
    int Row = blockIdx.y * TILE_DIM + threadIdx.y;
    int Col = blockIdx.x * TILE_DIM + threadIdx.x;
    __shared__ double As[TILE_DIM][TILE_DIM];
    __shared__ double Bs[TILE_DIM][TILE_DIM];

	for (int i = 0; i < (TILE_DIM + ACols - 1) / TILE_DIM ; ++i) {

		if (i * TILE_DIM + threadIdx.x < ACols && Row < ARows)   
			As[threadIdx.y][threadIdx.x] = A[Row * ACols + i * TILE_DIM + threadIdx.x];
		else                                                   
			As[threadIdx.y][threadIdx.x] = 0.0;

		if (i * TILE_DIM + threadIdx.y < ACols && Col < BCols)   
			Bs[threadIdx.y][threadIdx.x] = B[(i * TILE_DIM + threadIdx.y) * BCols + Col];
		else                                                   
			Bs[threadIdx.y][threadIdx.x] = 0.0;
		__syncthreads();

		for (int j = 0; j < TILE_DIM; ++j) 
			CValue = CValue + As[threadIdx.y][j] * Bs[j][threadIdx.x];
		__syncthreads();
    }

    if (Row < ARows && Col < BCols) 
		C[((blockIdx.y * blockDim.y + threadIdx.y) * BCols) + (blockIdx.x * blockDim.x) + threadIdx.x] = CValue;
}

__global__ void reduce_memberships_kernel(double * memberships, double * odata, int N)
{
	extern __shared__ double sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x*blockDim.x + tid;
	int gridSize = blockDim.x*gridDim.x;
	double temp = 0.0;

	while(i < N){
		temp = temp + memberships[i];
		i += gridSize;
	}
	sdata[tid] = temp;
	__syncthreads();

	//if (blockDim.x > 511){
	//	if (tid < 256)
	//		sdata[tid] = sdata[tid] + sdata[tid+256];
	//	__syncthreads();
	//}

	if (blockDim.x > 255){
		if (tid < 128) sdata[tid] = sdata[tid] + sdata[tid+128];
		__syncthreads();
	}

	if (blockDim.x > 127){
		if (tid < 64) sdata[tid] = sdata[tid] + sdata[tid+64];
		__syncthreads();
	}

	if (tid < 32) sdata[tid] = sdata[tid] + sdata[tid + 32];
	if (tid < 16) sdata[tid] = sdata[tid] + sdata[tid + 16];
	if (tid < 8) sdata[tid] = sdata[tid] + sdata[tid + 8];
	if (tid < 4) sdata[tid] = sdata[tid] + sdata[tid + 4];
	if (tid < 2) sdata[tid] = sdata[tid] + sdata[tid + 2];

	if (tid == 0) odata[blockIdx.x] = sdata[0] + sdata[1];
}

__global__ void reduce_centroids_kernel
	(double * points, double * memberships, double * odata, int N)
{
	extern __shared__ double sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x*blockDim.x + tid;
	int gridSize = blockDim.x*gridDim.x;
	double temp = 0.0;
	
	while(i < N){
		temp = temp + points[i] * memberships[i];
		i += gridSize;
	}
	sdata[tid] = temp;
	__syncthreads();

	if (tid < 128) 
		sdata[tid] = sdata[tid] + sdata[tid+128];
	__syncthreads();

	if (tid < 64) 
		sdata[tid] = sdata[tid] + sdata[tid+64];
	__syncthreads();

	if (tid < 32) sdata[tid] = sdata[tid] + sdata[tid + 32];
	if (tid < 16) sdata[tid] = sdata[tid] + sdata[tid + 16];
	if (tid < 8) sdata[tid] = sdata[tid] + sdata[tid + 8];
	if (tid < 4) sdata[tid] = sdata[tid] + sdata[tid + 4];
	if (tid < 2) sdata[tid] = sdata[tid] + sdata[tid + 2];

	if (tid == 0) odata[blockIdx.x] = sdata[0] + sdata[1];
}

__global__ void calculate_new_centroids(double * centroids, double * memberships)
{
	int cid = blockIdx.x*blockDim.x + threadIdx.x;
	centroids[cid] = centroids[cid] / memberships[blockIdx.x];
}

__host__ void calculate_new_centroids(double * points, double * memberships, double * newCentroids, int N, int D, int K)
{
	int i, j, k;
	double * pPoints = points;
	double * pMemberships = memberships;
	double * pCentroids;
	double * sum = new double[K]();
	memset(newCentroids, 0, K*D*sizeof(double));

	for (i = 0; i < N; ++i, pMemberships += K, pPoints += D){
		pCentroids = newCentroids;

		for (j = 0; j < K; ++j, pCentroids += D){
			sum[j] = sum[j] + pMemberships[j];
			
			for (k = 0; k < D; ++k)
				pCentroids[k] = pCentroids[k] + pMemberships[j]*pPoints[k];
		}
	}
	pCentroids = newCentroids;

	for (i = 0; i < K; ++i, pCentroids += D)
		for (j = 0; j < D; ++j)
			pCentroids[j] = pCentroids[j] / sum[i];
}

__global__ void check_convergence(double * centroids, double * newCentroids, double epsilon, int * flag)
{
	flag[0] = 0;

	for (int i = 0; i < blockDim.x; ++i){
		if (fabs(centroids[i] - newCentroids[i]) >= epsilon) return;
	}
	flag[0] = 1;
}

__host__ double * FKM_GPU(FILE * f, FKM & G, int block_size, int stop_iter, int mode)
{
#pragma region Declare common variables
	int i, j, k, x, y, z, w;
	int DBL_SIZE = sizeof(double);
	
	int points_size = G.N * G.D * DBL_SIZE;
	int c_size = G.K * DBL_SIZE;
	int centroids_size = c_size * G.D;
	int uk_size = G.N * c_size;
	int sm_size = block_size * DBL_SIZE;
	int block_dsize = block_size<<2;
	int num_blocks = roundup(G.N, block_size);
	int num_cblocks = roundup(G.N, block_dsize);
	int u_size = num_cblocks * c_size;
	int tempC_size = num_cblocks * centroids_size;
	double t1 = 0.0, t2 = 0.0, t3 = 0.0, t4;
	
	TimingCPU tmr_CPU;
	TimingGPU tmr_GPU;

	dim3 dimBlock(TILE_DIM, TILE_DIM);
	dim3 dimGrid(roundup(G.D, dimBlock.x), roundup(G.K, dimBlock.y));

	double alpha, beta;
	double * p1;
	double * p2;
	double * p3;
#pragma endregion

#pragma region Declare device memories
	double * d_points;
	double * d_pointsT;
	double * d_centroids;
	double * d_memberships;
	double * d_membershipsT;
	double * d_u;
	double * d_sumU;
	double * d_tempC;
	int * d_stop;
#pragma endregion

#pragma region Declare host pinned memories
	double * p_points;
	double * p_centroids;
	double * p_memberships;
	double * p_u;
	double * p_sumU;
	double * p_tempC;
	int * p_stop;
#pragma endregion

#pragma region Malloc device
	CudaSafeCall(hipMalloc(&d_points, points_size));
	CudaSafeCall(hipMalloc(&d_pointsT, points_size));
	CudaSafeCall(hipMalloc(&d_centroids, centroids_size));
	CudaSafeCall(hipMalloc(&d_memberships, uk_size));
	CudaSafeCall(hipMalloc(&d_membershipsT, uk_size));
	CudaSafeCall(hipMalloc(&d_u, u_size));
	CudaSafeCall(hipMalloc(&d_sumU, c_size));
	CudaSafeCall(hipMalloc(&d_tempC, tempC_size));
	CudaSafeCall(hipMalloc(&d_stop, sizeof(int)));
#pragma endregion

#pragma region Malloc host
	CudaSafeCall(hipHostMalloc(&p_points, points_size));
	CudaSafeCall(hipHostMalloc(&p_centroids, centroids_size));
	CudaSafeCall(hipHostMalloc(&p_memberships, uk_size));
	CudaSafeCall(hipHostMalloc(&p_u, u_size));
	CudaSafeCall(hipHostMalloc(&p_sumU, c_size));
	CudaSafeCall(hipHostMalloc(&p_tempC, tempC_size));
	CudaSafeCall(hipHostMalloc(&p_stop, sizeof(int)));
#pragma endregion

#pragma region Memories copy
	memcpy(p_points, G.points, points_size);
	memcpy(p_centroids, G.centroids, centroids_size);
	CudaSafeCall(hipMemcpy(d_points, p_points, points_size, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_centroids, p_centroids, centroids_size, hipMemcpyHostToDevice));  
#pragma endregion

#pragma region Declare cuda streams and transpose points
	hipblasHandle_t handle;
	hipStream_t * streams = new hipStream_t[NSTREAM];

	if (mode == 1){
		for (i = 0; i < NSTREAM; ++i)
			hipStreamCreate(&streams[i]);

		CublasSafeCall(hipblasCreate(&handle));
		alpha = 1.;
		beta  = 0.;
		tmr_GPU.StartCounter();
		CublasSafeCall(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, G.N, G.D,
			&alpha, d_points, G.D, &beta, d_points, G.D, d_pointsT, G.N)); 
		t2 = t2 + tmr_GPU.GetCounter();
	}
#pragma endregion

#pragma region Main loop
	for (i = 0; i< G.max_iter; ++i){
#pragma region  Updating memberships by GPU
		tmr_GPU.StartCounter();
		update_memberships_kernel<<<num_blocks, block_size>>>
			(d_points, d_centroids,d_memberships, G.N, G.D, G.K, G.fuzzifier);
		//CudaCheckError();
		t1 = t1 + tmr_GPU.GetCounter();
#pragma endregion

		if (mode == 1){
#pragma region Transpose memberships
			alpha = 1.;
			beta  = 0.;
			tmr_GPU.StartCounter();
			CublasSafeCall(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, G.N, G.K,
				&alpha, d_memberships, G.K, &beta, d_memberships, G.K, d_membershipsT, G.N)); 
			t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

#pragma region Reduce centroids by GPU
			tmr_GPU.StartCounter();
			for (j = 0, x = 0, y = 0, z = 0; j < G.K; ++j, x += G.N, y += num_cblocks){
				reduce_memberships_kernel<<<num_cblocks, block_size, sm_size, streams[0]>>>
					(d_membershipsT + x, d_u + y, G.N);

				for (k = 0, w = 0; k < G.D; ++k, w += G.N, z += num_cblocks){
					reduce_centroids_kernel<<<num_cblocks, block_size, sm_size, streams[k % (NSTREAM-1)+1]>>>
							(d_pointsT + w, d_membershipsT + x, d_tempC + z, G.N);
				}
			}
			t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

#pragma region Reduce block sums by CPU
			if (num_cblocks > 1){
				tmr_GPU.StartCounter();
				CudaSafeCall(hipMemcpyAsync(p_u, d_u, u_size, hipMemcpyDeviceToHost));
				CudaSafeCall(hipMemcpyAsync(p_tempC, d_tempC, tempC_size, hipMemcpyDeviceToHost));
				t2 = t2 + tmr_GPU.GetCounter();
				tmr_CPU.start();

				for (j = 0, p1 = p_u, p2 = p_tempC, p3 = p_centroids; j < G.K; ++j, p1 += num_cblocks, p3 += G.D){
					p_sumU[j] = 0.0;

					for (x = 0; x < num_cblocks; ++x)
						p_sumU[j] = p_sumU[j] + p1[x];

					for (x = 0; x < G.D; ++x, p2 += num_cblocks){
						p3[x] = 0.0;

						for (y = 0; y < num_cblocks; ++y){
							p3[x] = p3[x] + p2[y];
						}
					}
				}
				tmr_CPU.stop();
				t2 = t2 + tmr_CPU.elapsed();
				tmr_GPU.StartCounter();
				CudaSafeCall(hipMemcpyAsync(d_u, p_sumU, c_size, hipMemcpyHostToDevice));
				CudaSafeCall(hipMemcpyAsync(d_tempC, p_centroids, centroids_size, hipMemcpyHostToDevice));
				t2 = t2 + tmr_GPU.GetCounter();
			}
#pragma endregion

#pragma region Calculate centroids by GPU
			tmr_GPU.StartCounter();
			calculate_new_centroids<<<G.K, G.D>>>(d_tempC, d_u);
			t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion
		}
		else{
#pragma region Calculate centroids by CPU
			tmr_GPU.StartCounter();
			CudaSafeCall(hipMemcpyAsync(p_memberships, d_memberships, uk_size, hipMemcpyDeviceToHost));
			t2 = t2 + tmr_GPU.GetCounter();
			tmr_CPU.start();
			calculate_new_centroids(p_points, p_memberships, p_centroids, G.N, G.D, G.K);
			tmr_CPU.stop();
			t2 = t2 + tmr_CPU.elapsed();
			tmr_GPU.StartCounter();
			CudaSafeCall(hipMemcpyAsync(d_tempC, p_centroids, centroids_size, hipMemcpyHostToDevice));
			t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion
		}
#pragma region Check convergence
		tmr_GPU.StartCounter();
		check_convergence<<<1, G.K*G.D>>>(d_centroids, d_tempC, G.epsilon, d_stop);
		CudaSafeCall(hipMemcpyAsync(p_stop, d_stop, sizeof(int), hipMemcpyDeviceToHost));
		t3 = t3 + tmr_GPU.GetCounter();
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(d_centroids, d_tempC, centroids_size, hipMemcpyDeviceToDevice));
		t2 = t2 + tmr_GPU.GetCounter();

		if ((p_stop[0] == 1 && (stop_iter == INT_MAX || i==stop_iter)) || i==stop_iter)
			break;
#pragma endregion
	}
	if (i == G.max_iter) i--;
#pragma endregion
	
#pragma region Copying device back to host
	tmr_GPU.StartCounter();
	CudaSafeCall(hipMemcpyAsync(p_centroids, d_centroids, centroids_size, hipMemcpyDeviceToHost));
	t4 = tmr_GPU.GetCounter();
#pragma endregion

#pragma region Write results to files
	Util::write<double>(p_centroids, G.K, G.D, G.path + "centroids.GPU.txt");
	Util::print_times(f, t1, t2, t3, i+1);
#pragma endregion

#pragma region Cuda free device memories
	hipFree(d_points);
	hipFree(d_pointsT);
	hipFree(d_centroids);
	hipFree(d_memberships);
	hipFree(d_membershipsT);
	hipFree(d_u);
	hipFree(d_sumU);
	hipFree(d_tempC);
	hipFree(d_stop);
#pragma endregion

#pragma region Cuda free host pinned memories
	hipHostFree(p_points);
	hipHostFree(p_centroids);
	hipHostFree(p_memberships);
	hipHostFree(p_u);
	hipHostFree(p_sumU);
	hipHostFree(p_tempC);
	hipHostFree(p_stop);
#pragma endregion

#pragma region Get total time and last iteration index
	double * rs = new double[2];
	rs[0] = t1 + t2 + t3 + t4;
	rs[1] = (double)i;
#pragma endregion

#pragma region CublasDestroy, CudaStreamDestroy, and DeviceReset
	if (mode == 1)
	{
		CublasSafeCall(hipblasDestroy(handle));

		for (i = 0; i < NSTREAM; ++i)
			hipStreamDestroy(streams[i]);
	}
	
	hipDeviceReset();
#pragma endregion
	
	return rs;
}