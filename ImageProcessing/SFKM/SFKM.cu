#include "hip/hip_runtime.h"
#include "SFKM.cuh"
#include "Util.h"
#define DIM_MAX 100
#define NSTREAM 5

inline __host__ int roundup(int x, int y)
{
	return 1 + (x-1)/y;
}

__global__ void update_memberships_kernel_v1a(
	double * points, double * centroids, double * memberships, 
	int N, int D, int K, double fuzzifier)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= N) return;
	int i, j;
	double * pMemberships = memberships + idx*K;
	double * pCentroids = centroids;
	double X[DIM_MAX];
	double f = 1. / (fuzzifier - 1.);
	double diff, temp, sum = 0.;

	for (i = 0, j = idx*D; i < D; ++i, ++j) X[i] = points[j];

	for (i = 0; i < K; ++i) pMemberships[i] = 0.;

	for (i = 0; i < K; ++i, pCentroids += D){
		diff = 0.;

		for (j = 0; j < D; ++j){
			temp = X[j] - pCentroids[j];
			diff = diff + temp*temp;
		}

		if (diff == 0.){ 
			pMemberships[i] = 1.;
			return;
		}
		diff = pow(diff, f);
		pMemberships[i] = diff;
		sum = sum + 1. / diff;
	}

	for (i = 0; i < K; ++i){
		pMemberships[i] = pow(pMemberships[i]*sum, -fuzzifier);
	}
}

__global__ void update_memberships_kernel_v1b(
	double * points, double * centroids, double * memberships, 
	int N, int D, int K, double fuzzifier)
{
	extern __shared__ double C[];
	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + tid;

	if (tid < K*D){
		C[tid] = centroids[tid];
	}
	__syncthreads();

	if (idx >= N) return;
	int i, j;
	double * pMemberships = memberships + idx*K;
	double * pCentroids = C;
	double X[DIM_MAX];
	double f = 1. / (fuzzifier - 1.);
	double diff, temp, sum = 0.;

	for (i = 0, j = idx*D; i < D; ++i, ++j) X[i] = points[j];

	for (i = 0; i < K; ++i) pMemberships[i] = 0.;

	for (i = 0; i < K; ++i, pCentroids += D){
		diff = 0.;

		for (j = 0; j < D; ++j){
			temp = X[j] - pCentroids[j];
			diff = diff + temp*temp;
		}

		if (diff == 0.){ 
			pMemberships[i] = 1.;
			return;
		}
		diff = pow(diff, f);
		pMemberships[i] = diff;
		sum = sum + 1. / diff;
	}

	for (i = 0; i < K; ++i){
		pMemberships[i] = pow(pMemberships[i]*sum, -fuzzifier);
	}
}

__global__ void update_memberships_kernel_v1c(
	double * points, double * centroids, double * memberships, 
	int N, int D, int K, int step, double fuzzifier)
{
	extern __shared__ double C[];

	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + tid;
	int i, j;
	
	for (i = 0, j = K*D, tid *= step; tid < j && i < step; ++i, ++tid){
		C[tid] = centroids[tid];
	}
	__syncthreads();

	if (idx >= N) return;
	double * pMemberships = memberships + idx*K;
	double * pCentroids = C;
	double X[DIM_MAX];
	double f = 1. / (fuzzifier - 1.);
	double diff, temp, sum = 0.;

	for (i = 0, j = idx*D; i < D; ++i, ++j) X[i] = points[j];

	for (i = 0; i < K; ++i) pMemberships[i] = 0.;

	for (i = 0; i < K; ++i, pCentroids += D){
		diff = 0.;

		for (j = 0; j < D; ++j){
			temp = X[j] - pCentroids[j];
			diff = diff + temp*temp;
		}

		if (diff == 0.){ 
			pMemberships[i] = 1.;
			return;
		}
		diff = pow(diff, f);
		pMemberships[i] = diff;
		sum = sum + 1. / diff;
	}

	for (i = 0; i < K; ++i){
		pMemberships[i] = pow(pMemberships[i]*sum, -fuzzifier);
	}
}

__global__ void reduce_memberships_kernel(double * memberships, double * sumU, int N)
{
	extern __shared__ double sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x*blockDim.x + tid;
	int gridSize = blockDim.x*gridDim.x;
	double temp = 0.0;

	while(i < N){
		temp = temp + memberships[i];
		i += gridSize;
	}
	sdata[tid] = temp;
	__syncthreads();

	if (blockDim.x > 255){
		if (tid < 128) sdata[tid] = sdata[tid] + sdata[tid+128];
		__syncthreads();
	}

	if (blockDim.x > 127){
		if (tid < 64) sdata[tid] = sdata[tid] + sdata[tid+64];
		__syncthreads();
	}

	if (tid < 32) sdata[tid] = sdata[tid] + sdata[tid + 32];
	if (tid < 16) sdata[tid] = sdata[tid] + sdata[tid + 16];
	if (tid < 8) sdata[tid] = sdata[tid] + sdata[tid + 8];
	if (tid < 4) sdata[tid] = sdata[tid] + sdata[tid + 4];
	if (tid < 2) sdata[tid] = sdata[tid] + sdata[tid + 2];

	if (tid == 0) sumU[blockIdx.x] = sdata[0] + sdata[1];
}

__global__ void reduce_centroids_kernel
	(double * points, double * memberships, double * sumC, int N)
{
	extern __shared__ double sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x*blockDim.x + tid;
	int gridSize = blockDim.x*gridDim.x;
	double temp = 0.0;
	
	while(i < N){
		temp = temp + points[i] * memberships[i];
		i += gridSize;
	}
	sdata[tid] = temp;
	__syncthreads();

	if (tid < 128) 
		sdata[tid] = sdata[tid] + sdata[tid+128];
	__syncthreads();

	if (tid < 64) 
		sdata[tid] = sdata[tid] + sdata[tid+64];
	__syncthreads();

	if (tid < 32) sdata[tid] = sdata[tid] + sdata[tid + 32];
	if (tid < 16) sdata[tid] = sdata[tid] + sdata[tid + 16];
	if (tid < 8) sdata[tid] = sdata[tid] + sdata[tid + 8];
	if (tid < 4) sdata[tid] = sdata[tid] + sdata[tid + 4];
	if (tid < 2) sdata[tid] = sdata[tid] + sdata[tid + 2];

	if (tid == 0) sumC[blockIdx.x] = sdata[0] + sdata[1];
}

__host__ void reduce_centroids
	(double * centroids, double * sumC, double * sumU, int num_reduction_blocks, int D, int K)
{
	double * p_centroids = centroids;
	double * p_sumU = sumU;
	double * p_sumC = sumC;

	double u;
	int i, j, k;

	for (i = 0; i < K; ++i){
		u = 0.0;

		for (j = 0; j < num_reduction_blocks; ++j)
			u = u + p_sumU[j];

		for (j = 0; j < D; ++j){
			p_centroids[j] = 0.0;

			for (k = 0; k < num_reduction_blocks; ++k){
				p_centroids[j] = p_centroids[j] + p_sumC[k];
			}
			p_centroids[j] = p_centroids[j] /u;
			p_sumC += num_reduction_blocks;
		}
		p_sumU += num_reduction_blocks;
		p_centroids += D;
	}
}

__global__ void calculate_new_centroids(double * centroids, double * memberships)
{
	int cid = blockIdx.x*blockDim.x + threadIdx.x;
	centroids[cid] = centroids[cid] / memberships[blockIdx.x];
}

__host__ void calculate_new_centroids(double * points, double * memberships, double * newCentroids, int N, int D, int K)
{
	int i, j, k;
	double * pPoints = points;
	double * pMemberships = memberships;
	double * pCentroids;
	double * sum = new double[K]();
	memset(newCentroids, 0, K*D*sizeof(double));

	for (i = 0; i < N; ++i, pMemberships += K, pPoints += D){
		pCentroids = newCentroids;

		for (j = 0; j < K; ++j, pCentroids += D){
			sum[j] = sum[j] + pMemberships[j];
			
			for (k = 0; k < D; ++k)
				pCentroids[k] = pCentroids[k] + pMemberships[j]*pPoints[k];
		}
	}
	pCentroids = newCentroids;

	for (i = 0; i < K; ++i, pCentroids += D)
		for (j = 0; j < D; ++j)
			pCentroids[j] = pCentroids[j] / sum[i];
}

__global__ void check_convergence(double * centroids, double * newCentroids, bool * flag, double epsilon)
{
	int cid = blockDim.x * blockIdx.x + threadIdx.x;
	flag[0] = fabs(centroids[cid] - newCentroids[cid]) >= epsilon;
	//__threadfence();
	/*flag[0] = false;
	int n = blockDim.x;

	for (int i = 0; i < n; ++i){
		if (fabs(centroids[i] - newCentroids[i]) >= epsilon){
			flag[0] = true;
			return;
		}
	}*/
}

__host__ double * FKM_GPU_v1a(FILE * f, FKM & G, int block_size, int stop_iter)
{
#pragma region Declare common variables
	int i;
	int DBL_SIZE = sizeof(double);
	int flag_size = sizeof(bool);

	int points_size = G.N * G.D * DBL_SIZE;
	int centroids_size = G.K * G.D * DBL_SIZE;
	int memberships_size = G.N * G.K* DBL_SIZE;

	int num_blocks = roundup(G.N, block_size);

	double t1 = 0.0, t2 = 0.0, t3 = 0.0;

	TimingCPU tmr_CPU;
	TimingGPU tmr_GPU;
#pragma endregion

#pragma region Declare device memories
	bool * d_flags;
	double * d_points;
	double * d_centroids;
	double * d_newCentroids;
	double * d_memberships;
#pragma endregion

#pragma region Declare host pinned memories
	bool * p_flags;
	double * p_points;
	double * p_centroids;
	double * p_memberships;
#pragma endregion

#pragma region Malloc device
	CudaSafeCall(hipMalloc(&d_flags, flag_size));
	CudaSafeCall(hipMalloc(&d_points, points_size));
	CudaSafeCall(hipMalloc(&d_centroids, centroids_size));
	CudaSafeCall(hipMalloc(&d_newCentroids, centroids_size));
	CudaSafeCall(hipMalloc(&d_memberships, memberships_size));
#pragma endregion

#pragma region Malloc host
	CudaSafeCall(hipHostMalloc(&p_flags, flag_size));
	CudaSafeCall(hipHostMalloc(&p_points, points_size));
	CudaSafeCall(hipHostMalloc(&p_centroids, centroids_size));
	CudaSafeCall(hipHostMalloc(&p_memberships, memberships_size));
#pragma endregion

#pragma region Memories copy
	memcpy(p_points, G.points, points_size);
	memcpy(p_centroids, G.centroids, centroids_size);
	CudaSafeCall(hipMemcpy(d_points, p_points, points_size, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_centroids, p_centroids, centroids_size, hipMemcpyHostToDevice));  
#pragma endregion

#pragma region Main loop
	for (i = 0; i< G.max_iter; ++i){
#pragma region  Update memberships by GPU
		tmr_GPU.StartCounter();
		update_memberships_kernel_v1a<<<num_blocks, block_size>>>
			(d_points, d_centroids,d_memberships, G.N, G.D, G.K, G.fuzzifier);
		//CudaCheckError();
		t1 = t1 + tmr_GPU.GetCounter();
#pragma endregion
		
#pragma region Calculate new centroids by CPU
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(p_memberships, d_memberships, memberships_size, hipMemcpyDeviceToHost));
		t2 = t2 + tmr_GPU.GetCounter();
		tmr_CPU.start();
		calculate_new_centroids(p_points, p_memberships, p_centroids, G.N, G.D, G.K);
		tmr_CPU.stop();
		t2 = t2 + tmr_CPU.elapsed();
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(d_newCentroids, p_centroids, centroids_size, hipMemcpyHostToDevice));
		t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

#pragma region Getting and checking stop-condition
		tmr_GPU.StartCounter();
		check_convergence<<<G.K, G.D>>>(d_centroids, d_newCentroids, d_flags, G.epsilon);
		CudaSafeCall(hipMemcpyAsync(p_flags, d_flags, flag_size, hipMemcpyDeviceToHost));
		t3 = t3 + tmr_GPU.GetCounter();
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(d_centroids, d_newCentroids, centroids_size, hipMemcpyDeviceToDevice));
		t2 = t2 + tmr_GPU.GetCounter();

		if ((!p_flags[0] && (stop_iter == INT_MAX || i==stop_iter)) || i==stop_iter)
			break;
#pragma endregion
	}
	if (i == G.max_iter) i--;
#pragma endregion

#pragma region Writing results to files
	Util::write<double>(p_centroids, G.K, G.D, G.path + "centroids.GPU.txt");
	Util::write<double>(p_memberships, G.N, G.K, G.path + "memberships.GPU.txt");
	Util::print_times(f, t1, t2, t3, i+1);
#pragma endregion

#pragma region Cuda free device memories
	hipFree(d_flags);
	hipFree(d_points);
	hipFree(d_centroids);
	hipFree(d_newCentroids);
	hipFree(d_memberships);
#pragma endregion

#pragma region Cuda free host pinned memories
	hipHostFree(p_flags);
	hipHostFree(p_points);
	hipHostFree(p_centroids);
	hipHostFree(p_memberships);
#pragma endregion

#pragma region Get total time and last iteration index
	double * rs = new double[2];
	rs[0] = t1 + t2 + t3;
	rs[1] = (double)i;
#pragma endregion

	hipDeviceReset();
	return rs;
}

__host__ double * FKM_GPU_v1b(FILE * f, FKM & G, int block_size, int stop_iter)
{
#pragma region Declare common variables
	int i;
	int DBL_SIZE = sizeof(double);
	int flag_size = sizeof(bool);

	int points_size = G.N * G.D * DBL_SIZE;
	int centroids_size = G.K * G.D * DBL_SIZE;
	int memberships_size = G.N * G.K * DBL_SIZE;

	int num_blocks = roundup(G.N, block_size);

	double t1 = 0.0, t2 = 0.0, t3 = 0.0;

	TimingCPU tmr_CPU;
	TimingGPU tmr_GPU;
#pragma endregion

#pragma region Declare device memories
	bool * d_flags;
	double * d_points;
	double * d_centroids;
	double * d_newCentroids;
	double * d_memberships;
#pragma endregion

#pragma region Declare host pinned memories
	bool * p_flags;
	double * p_points;
	double * p_centroids;
	double * p_memberships;
#pragma endregion

#pragma region Malloc device
	CudaSafeCall(hipMalloc(&d_flags, flag_size));
	CudaSafeCall(hipMalloc(&d_points, points_size));
	CudaSafeCall(hipMalloc(&d_centroids, centroids_size));
	CudaSafeCall(hipMalloc(&d_newCentroids, centroids_size));
	CudaSafeCall(hipMalloc(&d_memberships, memberships_size));
#pragma endregion

#pragma region Malloc host
	CudaSafeCall(hipHostMalloc(&p_flags, flag_size));
	CudaSafeCall(hipHostMalloc(&p_points, points_size));
	CudaSafeCall(hipHostMalloc(&p_centroids, centroids_size));
	CudaSafeCall(hipHostMalloc(&p_memberships, memberships_size));
#pragma endregion

#pragma region Memories copy
	memcpy(p_points, G.points, points_size);
	memcpy(p_centroids, G.centroids, centroids_size);
	CudaSafeCall(hipMemcpy(d_points, p_points, points_size, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_centroids, p_centroids, centroids_size, hipMemcpyHostToDevice));  
#pragma endregion

#pragma region Main loop
	for (i = 0; i< G.max_iter; ++i){
#pragma region  Update memberships by GPU
		tmr_GPU.StartCounter();
		update_memberships_kernel_v1b<<<num_blocks, block_size, centroids_size>>>
			(d_points, d_centroids,d_memberships, G.N, G.D, G.K, G.fuzzifier);
		//CudaCheckError();
		t1 = t1 + tmr_GPU.GetCounter();
#pragma endregion
		
#pragma region Calculate new centroids by CPU
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(p_memberships, d_memberships, memberships_size, hipMemcpyDeviceToHost));
		t2 = t2 + tmr_GPU.GetCounter();
		tmr_CPU.start();
		calculate_new_centroids(p_points, p_memberships, p_centroids, G.N, G.D, G.K);
		tmr_CPU.stop();
		t2 = t2 + tmr_CPU.elapsed();
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(d_newCentroids, p_centroids, centroids_size, hipMemcpyHostToDevice));
		t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

#pragma region Getting and checking stop-condition
		tmr_GPU.StartCounter();
		check_convergence<<<G.K, G.D>>>(d_centroids, d_newCentroids, d_flags, G.epsilon);
		CudaSafeCall(hipMemcpyAsync(p_flags, d_flags, flag_size, hipMemcpyDeviceToHost));
		t3 = t3 + tmr_GPU.GetCounter();
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(d_centroids, d_newCentroids, centroids_size, hipMemcpyDeviceToDevice));
		t2 = t2 + tmr_GPU.GetCounter();

		if ((!p_flags[0] && (stop_iter == INT_MAX || i==stop_iter)) || i==stop_iter)
			break;
#pragma endregion
	}
	if (i == G.max_iter) i--;
#pragma endregion

#pragma region Writing results to files
	Util::write<double>(p_centroids, G.K, G.D, G.path + "centroids.GPU.txt");
	Util::write<double>(p_memberships, G.N, G.K, G.path + "memberships.GPU.txt");
	Util::print_times(f, t1, t2, t3, i+1);
#pragma endregion

#pragma region Cuda free device memories
	hipFree(d_flags);
	hipFree(d_points);
	hipFree(d_centroids);
	hipFree(d_newCentroids);
	hipFree(d_memberships);
#pragma endregion

#pragma region Cuda free host pinned memories
	hipHostFree(p_flags);
	hipHostFree(p_points);
	hipHostFree(p_centroids);
	hipHostFree(p_memberships);
#pragma endregion

#pragma region Get total time and last iteration index
	double * rs = new double[2];
	rs[0] = t1 + t2 + t3;
	rs[1] = (double)i;
#pragma endregion

	hipDeviceReset();
	return rs;
}

__host__ double * FKM_GPU_v1c(FILE * f, FKM & G, int block_size, int stop_iter, int step)
{
#pragma region Declare common variables
	int i;
	int DBL_SIZE = sizeof(double);
	int flag_size = sizeof(bool);

	int points_size = G.N * G.D * DBL_SIZE;
	int centroids_size = G.K * G.D * DBL_SIZE;
	int memberships_size = G.N * G.K * DBL_SIZE;

	int num_blocks = roundup(G.N, block_size);

	double t1 = 0.0, t2 = 0.0, t3 = 0.0;

	TimingCPU tmr_CPU;
	TimingGPU tmr_GPU;
#pragma endregion

#pragma region Declare device memories
	bool * d_flags;
	double * d_points;
	double * d_centroids;
	double * d_newCentroids;
	double * d_memberships;
#pragma endregion

#pragma region Declare host pinned memories
	bool * p_flags;
	double * p_points;
	double * p_centroids;
	double * p_memberships;
#pragma endregion

#pragma region Malloc device
	CudaSafeCall(hipMalloc(&d_flags, flag_size));
	CudaSafeCall(hipMalloc(&d_points, points_size));
	CudaSafeCall(hipMalloc(&d_centroids, centroids_size));
	CudaSafeCall(hipMalloc(&d_newCentroids, centroids_size));
	CudaSafeCall(hipMalloc(&d_memberships, memberships_size));
#pragma endregion

#pragma region Malloc host
	CudaSafeCall(hipHostMalloc(&p_flags, flag_size));
	CudaSafeCall(hipHostMalloc(&p_points, points_size));
	CudaSafeCall(hipHostMalloc(&p_centroids, centroids_size));
	CudaSafeCall(hipHostMalloc(&p_memberships, memberships_size));
#pragma endregion

#pragma region Memories copy
	memcpy(p_points, G.points, points_size);
	memcpy(p_centroids, G.centroids, centroids_size);
	CudaSafeCall(hipMemcpy(d_points, p_points, points_size, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_centroids, p_centroids, centroids_size, hipMemcpyHostToDevice));  
#pragma endregion

#pragma region Main loop
	for (i = 0; i< G.max_iter; ++i){
#pragma region  Update memberships by GPU
		tmr_GPU.StartCounter();
		update_memberships_kernel_v1c<<<num_blocks, block_size, centroids_size>>>
			(d_points, d_centroids,d_memberships, G.N, G.D, G.K, step, G.fuzzifier);
		//CudaCheckError();
		t1 = t1 + tmr_GPU.GetCounter();
#pragma endregion
		
#pragma region Calculate new centroids by CPU
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(p_memberships, d_memberships, memberships_size, hipMemcpyDeviceToHost));
		t2 = t2 + tmr_GPU.GetCounter();
		tmr_CPU.start();
		calculate_new_centroids(p_points, p_memberships, p_centroids, G.N, G.D, G.K);
		tmr_CPU.stop();
		t2 = t2 + tmr_CPU.elapsed();
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(d_newCentroids, p_centroids, centroids_size, hipMemcpyHostToDevice));
		t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

#pragma region Getting and checking stop-condition
		tmr_GPU.StartCounter();
		check_convergence<<<G.K, G.D>>>(d_centroids, d_newCentroids, d_flags, G.epsilon);
		CudaSafeCall(hipMemcpyAsync(p_flags, d_flags, flag_size, hipMemcpyDeviceToHost));
		t3 = t3 + tmr_GPU.GetCounter();
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(d_centroids, d_newCentroids, centroids_size, hipMemcpyDeviceToDevice));
		t2 = t2 + tmr_GPU.GetCounter();

		if ((!p_flags[0] && (stop_iter == INT_MAX || i==stop_iter)) || i==stop_iter)
			break;
#pragma endregion
	}
	if (i == G.max_iter) i--;
#pragma endregion

#pragma region Writing results to files
	Util::write<double>(p_centroids, G.K, G.D, G.path + "centroids.GPU.txt");
	Util::write<double>(p_memberships, G.N, G.K, G.path + "memberships.GPU.txt");
	Util::print_times(f, t1, t2, t3, i+1);
#pragma endregion

#pragma region Cuda free device memories
	hipFree(d_flags);
	hipFree(d_points);
	hipFree(d_centroids);
	hipFree(d_newCentroids);
	hipFree(d_memberships);
#pragma endregion

#pragma region Cuda free host pinned memories
	hipHostFree(p_flags);
	hipHostFree(p_points);
	hipHostFree(p_centroids);
	hipHostFree(p_memberships);
#pragma endregion

#pragma region Get total time and last iteration index
	double * rs = new double[2];
	rs[0] = t1 + t2 + t3;
	rs[1] = (double)i;
#pragma endregion

	hipDeviceReset();
	return rs;
}

__host__ double * FKM_GPU_v2a(FILE * f, FKM & G, int block_size, int stop_iter)
{
#pragma region Declare common variables
	int i, j, k;
	int DBL_SIZE = sizeof(double);
	int flag_size = sizeof(bool);

	int points_size = G.N * G.D * DBL_SIZE;
	int centroid_size = G.K * DBL_SIZE;
	int centroids_size = G.K * G.D * DBL_SIZE;
	int memberships_size = G.N * G.K * DBL_SIZE;
	
	int sm_size = block_size * DBL_SIZE;
	
	int num_blocks = roundup(G.N, block_size);
	int num_reduction_blocks;

	int reduction_block_size = block_size<<2;

	num_reduction_blocks = roundup(G.N, reduction_block_size);

	int sumU_size = num_reduction_blocks * centroid_size;
	int sumC_size = num_reduction_blocks * centroids_size;

	int offset;
	int offset_sumU;
	int offset_sumC;
	int offset_pointsT;

	TimingCPU tmr_CPU;
	TimingGPU tmr_GPU;

	double alpha, beta;
	double t1 = 0.0, t2 = 0.0, t3 = 0.0, t4;
#pragma endregion

#pragma region Declare device memories
	bool * d_flags;

	double * d_points;
	double * d_pointsT;

	double * d_centroids;

	double * d_memberships;
	double * d_membershipsT;

	double * d_sumU;
	double * d_sumC;
#pragma endregion

#pragma region Declare host pinned memories
	bool * p_flags;

	double * p_points;

	double * p_centroids;

	double * p_memberships;

	double * p_sumU;
	double * p_sumC;
#pragma endregion

#pragma region Malloc device
	CudaSafeCall(hipMalloc(&d_flags, flag_size));

	CudaSafeCall(hipMalloc(&d_points, points_size));
	CudaSafeCall(hipMalloc(&d_pointsT, points_size));

	CudaSafeCall(hipMalloc(&d_centroids, centroids_size));

	CudaSafeCall(hipMalloc(&d_memberships, memberships_size));
	CudaSafeCall(hipMalloc(&d_membershipsT, memberships_size));

	CudaSafeCall(hipMalloc(&d_sumU, sumU_size));
	CudaSafeCall(hipMalloc(&d_sumC, sumC_size));
#pragma endregion

#pragma region Malloc host
	CudaSafeCall(hipHostMalloc(&p_flags, flag_size));

	CudaSafeCall(hipHostMalloc(&p_points, points_size));

	CudaSafeCall(hipHostMalloc(&p_centroids, centroids_size));

	CudaSafeCall(hipHostMalloc(&p_memberships, memberships_size));

	CudaSafeCall(hipHostMalloc(&p_sumU, sumU_size));
	CudaSafeCall(hipHostMalloc(&p_sumC, sumC_size));

#pragma endregion

#pragma region Memories copy
	memcpy(p_points, G.points, points_size);
	memcpy(p_centroids, G.centroids, centroids_size);
	CudaSafeCall(hipMemcpy(d_points, p_points, points_size, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_centroids, p_centroids, centroids_size, hipMemcpyHostToDevice));  
#pragma endregion

#pragma region Declare cuda streams and transpose points
	hipblasHandle_t handle;
	hipStream_t * streams = new hipStream_t[NSTREAM];

	for (i = 0; i < NSTREAM; ++i)
		hipStreamCreate(&streams[i]);

	CublasSafeCall(hipblasCreate(&handle));
	alpha = 1.;
	beta  = 0.;
	tmr_GPU.StartCounter();
	CublasSafeCall(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, G.N, G.D,
		&alpha, d_points, G.D, &beta, d_points, G.D, d_pointsT, G.N)); 
	t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

#pragma region Main loop
	for (i = 0; i< G.max_iter; ++i){
#pragma region  Update memberships by GPU
		tmr_GPU.StartCounter();
		update_memberships_kernel_v1a<<<num_blocks, block_size>>>
			(d_points, d_centroids, d_memberships, G.N, G.D, G.K, G.fuzzifier);
		//CudaCheckError();
		t1 = t1 + tmr_GPU.GetCounter();
#pragma endregion
		
#pragma region Transpose memberships
		alpha = 1.;
		beta  = 0.;
		tmr_GPU.StartCounter();
		CublasSafeCall(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, G.N, G.K,
			&alpha, d_memberships, G.K, &beta, d_memberships, G.K, d_membershipsT, G.N)); 
		t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

#pragma region Reduce centroids by GPU
		tmr_GPU.StartCounter();
		offset = 0;
		offset_sumU = 0;
		offset_sumC = 0;

		for (j = 0; j < G.K; ++j){
			reduce_memberships_kernel<<<num_reduction_blocks, block_size, sm_size, streams[0]>>>
				(d_membershipsT + offset, d_sumU + offset_sumU, G.N);
			offset_pointsT = 0;

			for (k = 0; k < G.D; ++k){
				reduce_centroids_kernel<<<num_reduction_blocks, block_size, sm_size, streams[k % (NSTREAM-1)+1]>>>
						(d_pointsT + offset_pointsT, d_membershipsT + offset, d_sumC + offset_sumC, G.N);
				offset_pointsT += G.N;
				offset_sumC += num_reduction_blocks;
			}
			offset_sumU += num_reduction_blocks;
			offset += G.N;
		}
		t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

		if (num_reduction_blocks > 1){
#pragma region Reduce memberships and centroids block sums by CPU
			tmr_GPU.StartCounter();
			CudaSafeCall(hipMemcpyAsync(p_sumU, d_sumU, sumU_size, hipMemcpyDeviceToHost));
			CudaSafeCall(hipMemcpyAsync(p_sumC, d_sumC, sumC_size, hipMemcpyDeviceToHost));
			t2 = t2 + tmr_GPU.GetCounter();
			tmr_CPU.start();
			reduce_centroids(p_centroids, p_sumC, p_sumU, num_reduction_blocks, G.D, G.K);
			tmr_CPU.stop();
			t2 = t2 + tmr_CPU.elapsed();
			tmr_GPU.StartCounter();
			CudaSafeCall(hipMemcpyAsync(d_sumC, p_centroids, centroids_size, hipMemcpyHostToDevice));
			t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion
		}
		else{
#pragma region Calculate centroids by GPU
			tmr_GPU.StartCounter();
			calculate_new_centroids<<<G.K, G.D>>>(d_sumC, d_sumU);
			t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion
		}
#pragma region Getting and checking stop-condition
		tmr_GPU.StartCounter();
		check_convergence<<<G.K, G.D>>>(d_centroids, d_sumC, d_flags, G.epsilon);
		CudaSafeCall(hipMemcpyAsync(p_flags, d_flags, flag_size, hipMemcpyDeviceToHost));
		t3 = t3 + tmr_GPU.GetCounter();
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(d_centroids, d_sumC, centroids_size, hipMemcpyDeviceToDevice));
		t2 = t2 + tmr_GPU.GetCounter();

		if ((!p_flags[0] && (stop_iter == INT_MAX || i==stop_iter)) || i==stop_iter)
			break;
#pragma endregion
	}

	if (i == G.max_iter) i--;
#pragma endregion

#pragma region Copying device back to host
	tmr_GPU.StartCounter();
	CudaSafeCall(hipMemcpyAsync(p_centroids, d_centroids, centroids_size, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpyAsync(p_memberships, d_memberships, memberships_size, hipMemcpyDeviceToHost));
	t4 = tmr_GPU.GetCounter();
#pragma endregion

#pragma region Writing results to files
	Util::write<double>(p_centroids, G.K, G.D, G.path + "centroids.GPU.txt");
	Util::write<double>(p_memberships, G.N, G.K, G.path + "memberships.GPU.txt");
	Util::print_times(f, t1, t2, t3, i+1);
#pragma endregion

#pragma region Cuda free device memories
	hipFree(d_flags);

	hipFree(d_points);
	hipFree(d_pointsT);

	hipFree(d_centroids);

	hipFree(d_memberships);
	hipFree(d_membershipsT);

	hipFree(d_sumU);
	hipFree(d_sumC);
#pragma endregion

#pragma region Cuda free host pinned memories
	hipHostFree(p_flags);

	hipHostFree(p_points);

	hipHostFree(p_centroids);

	hipHostFree(p_memberships);

	hipHostFree(p_sumU);
	hipHostFree(p_sumC);
#pragma endregion

#pragma region Get total time and last iteration index
	double * rs = new double[2];
	rs[0] = t1 + t2 + t3 + t4;
	rs[1] = (double)i;
#pragma endregion

#pragma region CublasDestroy, CudaStreamDestroy, and DeviceReset
	CublasSafeCall(hipblasDestroy(handle));

	for (i = 0; i < NSTREAM; ++i)
		hipStreamDestroy(streams[i]);
	
	hipDeviceReset();
#pragma endregion
	
	return rs;
}

__host__ double * FKM_GPU_v2b(FILE * f, FKM & G, int block_size, int stop_iter)
{
#pragma region Declare common variables
	int i, j, k;
	int DBL_SIZE = sizeof(double);
	int flag_size = sizeof(bool);

	int points_size = G.N * G.D * DBL_SIZE;
	int centroid_size = G.K * DBL_SIZE;
	int centroids_size = G.K * G.D * DBL_SIZE;
	int memberships_size = G.N * G.K * DBL_SIZE;
	
	int sm_size = block_size * DBL_SIZE;
	
	int num_blocks = roundup(G.N, block_size);
	int num_reduction_blocks;

	int reduction_block_size = block_size<<2;

	num_reduction_blocks = roundup(G.N, reduction_block_size);

	int sumU_size = num_reduction_blocks * centroid_size;
	int sumC_size = num_reduction_blocks * centroids_size;

	int offset;
	int offset_sumU;
	int offset_sumC;
	int offset_pointsT;

	TimingCPU tmr_CPU;
	TimingGPU tmr_GPU;

	double alpha, beta;
	double t1 = 0.0, t2 = 0.0, t3 = 0.0, t4;
#pragma endregion

#pragma region Declare device memories
	bool * d_flags;

	double * d_points;
	double * d_pointsT;

	double * d_centroids;

	double * d_memberships;
	double * d_membershipsT;

	double * d_sumU;
	double * d_sumC;
#pragma endregion

#pragma region Declare host pinned memories
	bool * p_flags;

	double * p_points;

	double * p_centroids;

	double * p_memberships;

	double * p_sumU;
	double * p_sumC;
#pragma endregion

#pragma region Malloc device
	CudaSafeCall(hipMalloc(&d_flags, flag_size));

	CudaSafeCall(hipMalloc(&d_points, points_size));
	CudaSafeCall(hipMalloc(&d_pointsT, points_size));

	CudaSafeCall(hipMalloc(&d_centroids, centroids_size));

	CudaSafeCall(hipMalloc(&d_memberships, memberships_size));
	CudaSafeCall(hipMalloc(&d_membershipsT, memberships_size));

	CudaSafeCall(hipMalloc(&d_sumU, sumU_size));
	CudaSafeCall(hipMalloc(&d_sumC, sumC_size));
#pragma endregion

#pragma region Malloc host
	CudaSafeCall(hipHostMalloc(&p_flags, flag_size));

	CudaSafeCall(hipHostMalloc(&p_points, points_size));

	CudaSafeCall(hipHostMalloc(&p_centroids, centroids_size));

	CudaSafeCall(hipHostMalloc(&p_memberships, memberships_size));

	CudaSafeCall(hipHostMalloc(&p_sumU, sumU_size));
	CudaSafeCall(hipHostMalloc(&p_sumC, sumC_size));

#pragma endregion

#pragma region Memories copy
	memcpy(p_points, G.points, points_size);
	memcpy(p_centroids, G.centroids, centroids_size);
	CudaSafeCall(hipMemcpy(d_points, p_points, points_size, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_centroids, p_centroids, centroids_size, hipMemcpyHostToDevice));  
#pragma endregion

#pragma region Declare cuda streams and transpose points
	hipblasHandle_t handle;
	hipStream_t * streams = new hipStream_t[NSTREAM];

	for (i = 0; i < NSTREAM; ++i)
		hipStreamCreate(&streams[i]);

	CublasSafeCall(hipblasCreate(&handle));
	alpha = 1.;
	beta  = 0.;
	tmr_GPU.StartCounter();
	CublasSafeCall(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, G.N, G.D,
		&alpha, d_points, G.D, &beta, d_points, G.D, d_pointsT, G.N)); 
	t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

#pragma region Main loop
	for (i = 0; i< G.max_iter; ++i){
#pragma region  Update memberships by GPU
		tmr_GPU.StartCounter();
		update_memberships_kernel_v1b<<<num_blocks, block_size, centroids_size>>>
			(d_points, d_centroids, d_memberships, G.N, G.D, G.K, G.fuzzifier);
		//CudaCheckError();
		t1 = t1 + tmr_GPU.GetCounter();
#pragma endregion
		
#pragma region Transpose memberships
		alpha = 1.;
		beta  = 0.;
		tmr_GPU.StartCounter();
		CublasSafeCall(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, G.N, G.K,
			&alpha, d_memberships, G.K, &beta, d_memberships, G.K, d_membershipsT, G.N)); 
		t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

#pragma region Reduce centroids by GPU
		tmr_GPU.StartCounter();
		offset = 0;
		offset_sumU = 0;
		offset_sumC = 0;

		for (j = 0; j < G.K; ++j){
			reduce_memberships_kernel<<<num_reduction_blocks, block_size, sm_size, streams[0]>>>
				(d_membershipsT + offset, d_sumU + offset_sumU, G.N);
			offset_pointsT = 0;

			for (k = 0; k < G.D; ++k){
				reduce_centroids_kernel<<<num_reduction_blocks, block_size, sm_size, streams[k % (NSTREAM-1)+1]>>>
						(d_pointsT + offset_pointsT, d_membershipsT + offset, d_sumC + offset_sumC, G.N);
				offset_pointsT += G.N;
				offset_sumC += num_reduction_blocks;
			}
			offset_sumU += num_reduction_blocks;
			offset += G.N;
		}
		t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

		if (num_reduction_blocks > 1){
#pragma region Reduce memberships and centroids block sums by CPU
			tmr_GPU.StartCounter();
			CudaSafeCall(hipMemcpyAsync(p_sumU, d_sumU, sumU_size, hipMemcpyDeviceToHost));
			CudaSafeCall(hipMemcpyAsync(p_sumC, d_sumC, sumC_size, hipMemcpyDeviceToHost));
			t2 = t2 + tmr_GPU.GetCounter();
			tmr_CPU.start();
			reduce_centroids(p_centroids, p_sumC, p_sumU, num_reduction_blocks, G.D, G.K);
			tmr_CPU.stop();
			t2 = t2 + tmr_CPU.elapsed();
			tmr_GPU.StartCounter();
			CudaSafeCall(hipMemcpyAsync(d_sumC, p_centroids, centroids_size, hipMemcpyHostToDevice));
			t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion
		}
		else{
#pragma region Calculate centroids by GPU
			tmr_GPU.StartCounter();
			calculate_new_centroids<<<G.K, G.D>>>(d_sumC, d_sumU);
			t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion
		}
#pragma region Getting and checking stop-condition
		tmr_GPU.StartCounter();
		check_convergence<<<G.K, G.D>>>(d_centroids, d_sumC, d_flags, G.epsilon);
		CudaSafeCall(hipMemcpyAsync(p_flags, d_flags, flag_size, hipMemcpyDeviceToHost));
		t3 = t3 + tmr_GPU.GetCounter();
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(d_centroids, d_sumC, centroids_size, hipMemcpyDeviceToDevice));
		t2 = t2 + tmr_GPU.GetCounter();

		if ((!p_flags[0] && (stop_iter == INT_MAX || i==stop_iter)) || i==stop_iter)
			break;
#pragma endregion
	}

	if (i == G.max_iter) i--;
#pragma endregion

#pragma region Copying device back to host
	tmr_GPU.StartCounter();
	CudaSafeCall(hipMemcpyAsync(p_centroids, d_centroids, centroids_size, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpyAsync(p_memberships, d_memberships, memberships_size, hipMemcpyDeviceToHost));
	t4 = tmr_GPU.GetCounter();
#pragma endregion

#pragma region Writing results to files
	Util::write<double>(p_centroids, G.K, G.D, G.path + "centroids.GPU.txt");
	Util::write<double>(p_memberships, G.N, G.K, G.path + "memberships.GPU.txt");
	Util::print_times(f, t1, t2, t3, i+1);
#pragma endregion

#pragma region Cuda free device memories
	hipFree(d_flags);

	hipFree(d_points);
	hipFree(d_pointsT);

	hipFree(d_centroids);

	hipFree(d_memberships);
	hipFree(d_membershipsT);

	hipFree(d_sumU);
	hipFree(d_sumC);
#pragma endregion

#pragma region Cuda free host pinned memories
	hipHostFree(p_flags);

	hipHostFree(p_points);

	hipHostFree(p_centroids);

	hipHostFree(p_memberships);

	hipHostFree(p_sumU);
	hipHostFree(p_sumC);
#pragma endregion

#pragma region Get total time and last iteration index
	double * rs = new double[2];
	rs[0] = t1 + t2 + t3 + t4;
	rs[1] = (double)i;
#pragma endregion

#pragma region CublasDestroy, CudaStreamDestroy, and DeviceReset
	CublasSafeCall(hipblasDestroy(handle));

	for (i = 0; i < NSTREAM; ++i)
		hipStreamDestroy(streams[i]);
	
	hipDeviceReset();
#pragma endregion
	
	return rs;
}

__host__ double * FKM_GPU_v2c(FILE * f, FKM & G, int block_size, int stop_iter, int step)
{
#pragma region Declare common variables
	int i, j, k;
	int DBL_SIZE = sizeof(double);
	int flag_size = sizeof(bool);

	int points_size = G.N * G.D * DBL_SIZE;
	int centroid_size = G.K * DBL_SIZE;
	int centroids_size = G.K * G.D * DBL_SIZE;
	int memberships_size = G.N * G.K * DBL_SIZE;
	
	int sm_size = block_size * DBL_SIZE;
	
	int num_blocks = roundup(G.N, block_size);
	int num_reduction_blocks;

	int reduction_block_size = block_size<<2;

	num_reduction_blocks = roundup(G.N, reduction_block_size);

	int sumU_size = num_reduction_blocks * centroid_size;
	int sumC_size = num_reduction_blocks * centroids_size;

	int offset;
	int offset_sumU;
	int offset_sumC;
	int offset_pointsT;

	TimingCPU tmr_CPU;
	TimingGPU tmr_GPU;

	double alpha, beta;
	double t1 = 0.0, t2 = 0.0, t3 = 0.0, t4;
#pragma endregion

#pragma region Declare device memories
	bool * d_flags;

	double * d_points;
	double * d_pointsT;

	double * d_centroids;

	double * d_memberships;
	double * d_membershipsT;

	double * d_sumU;
	double * d_sumC;
#pragma endregion

#pragma region Declare host pinned memories
	bool * p_flags;

	double * p_points;

	double * p_centroids;

	double * p_memberships;

	double * p_sumU;
	double * p_sumC;
#pragma endregion

#pragma region Malloc device
	CudaSafeCall(hipMalloc(&d_flags, flag_size));

	CudaSafeCall(hipMalloc(&d_points, points_size));
	CudaSafeCall(hipMalloc(&d_pointsT, points_size));

	CudaSafeCall(hipMalloc(&d_centroids, centroids_size));

	CudaSafeCall(hipMalloc(&d_memberships, memberships_size));
	CudaSafeCall(hipMalloc(&d_membershipsT, memberships_size));

	CudaSafeCall(hipMalloc(&d_sumU, sumU_size));
	CudaSafeCall(hipMalloc(&d_sumC, sumC_size));
#pragma endregion

#pragma region Malloc host
	CudaSafeCall(hipHostMalloc(&p_flags, flag_size));

	CudaSafeCall(hipHostMalloc(&p_points, points_size));

	CudaSafeCall(hipHostMalloc(&p_centroids, centroids_size));

	CudaSafeCall(hipHostMalloc(&p_memberships, memberships_size));

	CudaSafeCall(hipHostMalloc(&p_sumU, sumU_size));
	CudaSafeCall(hipHostMalloc(&p_sumC, sumC_size));

#pragma endregion

#pragma region Memories copy
	memcpy(p_points, G.points, points_size);
	memcpy(p_centroids, G.centroids, centroids_size);
	CudaSafeCall(hipMemcpy(d_points, p_points, points_size, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_centroids, p_centroids, centroids_size, hipMemcpyHostToDevice));  
#pragma endregion

#pragma region Declare cuda streams and transpose points
	hipblasHandle_t handle;
	hipStream_t * streams = new hipStream_t[NSTREAM];

	for (i = 0; i < NSTREAM; ++i)
		hipStreamCreate(&streams[i]);

	CublasSafeCall(hipblasCreate(&handle));
	alpha = 1.;
	beta  = 0.;
	tmr_GPU.StartCounter();
	CublasSafeCall(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, G.N, G.D,
		&alpha, d_points, G.D, &beta, d_points, G.D, d_pointsT, G.N)); 
	t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

#pragma region Main loop
	for (i = 0; i< G.max_iter; ++i){
#pragma region  Update memberships by GPU
		tmr_GPU.StartCounter();
		update_memberships_kernel_v1c<<<num_blocks, block_size, centroids_size>>>
			(d_points, d_centroids, d_memberships, G.N, G.D, G.K, step, G.fuzzifier);
		//CudaCheckError();
		t1 = t1 + tmr_GPU.GetCounter();
#pragma endregion
		
#pragma region Transpose memberships
		alpha = 1.;
		beta  = 0.;
		tmr_GPU.StartCounter();
		CublasSafeCall(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, G.N, G.K,
			&alpha, d_memberships, G.K, &beta, d_memberships, G.K, d_membershipsT, G.N)); 
		t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

#pragma region Reduce centroids by GPU
		tmr_GPU.StartCounter();
		offset = 0;
		offset_sumU = 0;
		offset_sumC = 0;

		for (j = 0; j < G.K; ++j){
			reduce_memberships_kernel<<<num_reduction_blocks, block_size, sm_size, streams[0]>>>
				(d_membershipsT + offset, d_sumU + offset_sumU, G.N);
			offset_pointsT = 0;

			for (k = 0; k < G.D; ++k){
				reduce_centroids_kernel<<<num_reduction_blocks, block_size, sm_size, streams[k % (NSTREAM-1)+1]>>>
						(d_pointsT + offset_pointsT, d_membershipsT + offset, d_sumC + offset_sumC, G.N);
				offset_pointsT += G.N;
				offset_sumC += num_reduction_blocks;
			}
			offset_sumU += num_reduction_blocks;
			offset += G.N;
		}
		t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

		if (num_reduction_blocks > 1){
#pragma region Reduce memberships and centroids block sums by CPU
			tmr_GPU.StartCounter();
			CudaSafeCall(hipMemcpyAsync(p_sumU, d_sumU, sumU_size, hipMemcpyDeviceToHost));
			CudaSafeCall(hipMemcpyAsync(p_sumC, d_sumC, sumC_size, hipMemcpyDeviceToHost));
			t2 = t2 + tmr_GPU.GetCounter();
			tmr_CPU.start();
			reduce_centroids(p_centroids, p_sumC, p_sumU, num_reduction_blocks, G.D, G.K);
			tmr_CPU.stop();
			t2 = t2 + tmr_CPU.elapsed();
			tmr_GPU.StartCounter();
			CudaSafeCall(hipMemcpyAsync(d_sumC, p_centroids, centroids_size, hipMemcpyHostToDevice));
			t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion
		}
		else{
#pragma region Calculate centroids by GPU
			tmr_GPU.StartCounter();
			calculate_new_centroids<<<G.K, G.D>>>(d_sumC, d_sumU);
			t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion
		}
#pragma region Getting and checking stop-condition
		tmr_GPU.StartCounter();
		check_convergence<<<G.K, G.D>>>(d_centroids, d_sumC, d_flags, G.epsilon);
		CudaSafeCall(hipMemcpyAsync(p_flags, d_flags, flag_size, hipMemcpyDeviceToHost));
		t3 = t3 + tmr_GPU.GetCounter();
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(d_centroids, d_sumC, centroids_size, hipMemcpyDeviceToDevice));
		t2 = t2 + tmr_GPU.GetCounter();

		if ((!p_flags[0] && (stop_iter == INT_MAX || i==stop_iter)) || i==stop_iter)
			break;
#pragma endregion
	}

	if (i == G.max_iter) i--;
#pragma endregion

#pragma region Copying device back to host
	tmr_GPU.StartCounter();
	CudaSafeCall(hipMemcpyAsync(p_centroids, d_centroids, centroids_size, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpyAsync(p_memberships, d_memberships, memberships_size, hipMemcpyDeviceToHost));
	t4 = tmr_GPU.GetCounter();
#pragma endregion

#pragma region Writing results to files
	Util::write<double>(p_centroids, G.K, G.D, G.path + "centroids.GPU.txt");
	Util::write<double>(p_memberships, G.N, G.K, G.path + "memberships.GPU.txt");
	Util::print_times(f, t1, t2, t3, i+1);
#pragma endregion

#pragma region Cuda free device memories
	hipFree(d_flags);

	hipFree(d_points);
	hipFree(d_pointsT);

	hipFree(d_centroids);

	hipFree(d_memberships);
	hipFree(d_membershipsT);

	hipFree(d_sumU);
	hipFree(d_sumC);
#pragma endregion

#pragma region Cuda free host pinned memories
	hipHostFree(p_flags);

	hipHostFree(p_points);

	hipHostFree(p_centroids);

	hipHostFree(p_memberships);

	hipHostFree(p_sumU);
	hipHostFree(p_sumC);
#pragma endregion

#pragma region Get total time and last iteration index
	double * rs = new double[2];
	rs[0] = t1 + t2 + t3 + t4;
	rs[1] = (double)i;
#pragma endregion

#pragma region CublasDestroy, CudaStreamDestroy, and DeviceReset
	CublasSafeCall(hipblasDestroy(handle));

	for (i = 0; i < NSTREAM; ++i)
		hipStreamDestroy(streams[i]);
	
	hipDeviceReset();
#pragma endregion
	
	return rs;
}

__host__ double * FKM_GPU(FILE * f, FKM & G, int block_size, int stop_iter, int mode)
{
	int centroids_size = G.K*G.D;
	int step = roundup(centroids_size, block_size);

	if (mode == 1){
		if (step > 4)
			return FKM_GPU_v1a(f, G, block_size, stop_iter);
		else if (step == 1)
			return FKM_GPU_v1b(f, G, block_size, stop_iter);
		else
			return FKM_GPU_v1c(f, G, block_size, stop_iter, step);
	}
	else{
		if (step > 4)
			return FKM_GPU_v2a(f, G, block_size, stop_iter);
		else if (step == 1)
			return FKM_GPU_v2b(f, G, block_size, stop_iter);
		else
			return FKM_GPU_v2c(f, G, block_size, stop_iter, step);
	}
}