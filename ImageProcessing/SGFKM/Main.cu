#include "hip/hip_runtime.h"
#include "SGFKM.cuh"
#include "GFKM.h"
#include "TimingCPU.h"
#include <conio.h>

int main(int argc, char* argv[])
{
	std::string path = "D:\\Master\\ImageProcessing\\Data\\Synthetic\\";
	std::string fname = "960x512x100.dat";
	int M = 2;
	int max_iter = 1;
	int stop_iter = INT_MAX;
	int mode = 1;
	double epsilon = 1e-8;//numeric_limits<double>::epsilon();
	FILE * fp;

	if (argc > 2){
		path = std::string(argv[1]);
		fname = std::string(argv[2]);

		if (argc > 3){
			M = atoi(argv[3]);

			if (argc > 4){
				max_iter = atoi(argv[4]);

				if (argc > 5){
					epsilon = atof(argv[5]);

					if (argc > 6){
						mode = atoi(argv[6]);

						if (argc > 7){
							stop_iter = atoi(argv[7]) - 1;

							if (argc > 8){
								fp = fopen(argv[8], "a");
							}
						}
					}
				}
			}
		}
	}
	time_t rawtime;
	time ( &rawtime );
	GFKM G(path, fname, M);
	G.epsilon = epsilon;
	G.max_iter = max_iter;
	G.initialize_centroids();

	if(argc < 9) 
		fp = fopen("SGFKM.test.log", "a");
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# %s", ctime(&rawtime));
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp," %s%s\n N = %d, D = %d, K = %d, epsilon = %.0e\n", 
		path.c_str(), fname.c_str(), G.N, G.D, G.K, G.epsilon);
	fprintf(fp, " Mode: Calculating new centroids on %s\n", 
		mode == 1 ? "CPU" : mode == 2 ? "GPU (FKM)" : 
		mode == 3 ? "GPU (GFKM, counting sort)" : "GPU (GFKM, thrust stable sort by key)");
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# GPU running\n");
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	double * gpu_rs = GFKM_GPU(fp, G, 256, stop_iter, mode);
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# CPU running\n");
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	double * cpu_rs = G.run(fp, (int)gpu_rs[1]);
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# Total CPU time      : %9.2lf\n", cpu_rs[0]);
	fprintf(fp, "# Total GPU time      : %9.2lf\n", gpu_rs[0]);
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# Speedup             : %9.2lf\n", cpu_rs[0]/gpu_rs[0]);
	fclose(fp);
	printf(" Speedup: %.2lf\n", cpu_rs[0]/gpu_rs[0]);
	delete [] gpu_rs;
	delete [] cpu_rs;
	
	//getch();
    return 0;
}