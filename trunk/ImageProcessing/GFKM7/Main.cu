#include "hip/hip_runtime.h"
#include "GFKM.cuh"
#include "GFKM.h"
#include <string>

int main(int argc, char* argv[])
{
	string path = "D:\\Master\\ImageProcessing\\GFKM\\Data\\LenaPeppersBaboon\\";
	string fname = "LenaPeppersBaboon.txt";
	int M = 2;
	int max_iter = 300;
	double epsilon = 1e-10;//numeric_limits<double>::epsilon();
	int mode = 1;
	int stop_iter = 99;

	if (argc > 2){
		path = string(argv[1]);
		fname = string(argv[2]);
		cout << path << endl;

		if (argc > 3){
			M = atoi(argv[3]);

			if (argc > 4){
				max_iter = atoi(argv[4]);

				if (argc > 5){
					epsilon = atof(argv[5]);

					if (argc > 6){
						mode = atoi(argv[6]);

						if (argc > 7){
							stop_iter = atoi(argv[7]) - 1;
						}
					}
				}
			}
		}
	}
	time_t rawtime;
	time ( &rawtime );
	GFKM G(path, fname);
	G.epsilon = epsilon;
	G.max_iter = max_iter;
	G.M = M;
	G.initialize_centroids();
	FILE * fp = fopen("test.log","a");
	fprintf(fp, "%s", ctime(&rawtime));
	fprintf(fp,"%s\%s\n N = %d, D = %d, K = %d, M = %d, epsilon = %.0e\n Mode = %s\n", 
		path.c_str(), fname.c_str(), G.N, G.D, G.K, G.M, G.epsilon, 
		mode==0 ? "Updating centroids by CPU" : "Updating centroids by GPU");
	fprintf(fp, "----------------------------------------------------------------------------------\n");
	fprintf(fp, "GPU running\n");
	fprintf(fp, "----------------------------------------------------------------------------------\n");
	double * gpu_rs = GFKM_GPU(fp, G, 256, stop_iter, mode);
	G.J = 0.0;
	fprintf(fp, "----------------------------------------------------------------------------------\n");
	fprintf(fp, "CPU running\n");
	fprintf(fp, "----------------------------------------------------------------------------------\n");
	double * cpu_rs = G.run(fp, (int)gpu_rs[1]);
	fprintf(fp, "----------------------------------------------------------------------------------\n");
	fprintf(fp, "Total CPU time: %10.3lf\n", cpu_rs[0]);
	fprintf(fp, "Total GPU time: %10.3lf\n", gpu_rs[0]);
	fprintf(fp, "----------------------------------------------------------------------------------\n");
	fprintf(fp, "Total CPU time / Total GPU time: %.2lf\n", cpu_rs[0]/gpu_rs[0]);
	fprintf(fp, "----------------------------------------------------------------------------------\n");
	fclose(fp);
	delete [] gpu_rs;
	delete [] cpu_rs;
	//getch();
    return 0;
}