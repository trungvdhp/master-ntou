#include "hip/hip_runtime.h"
#include "GFKM.cuh"
#include "GFKM.h"
#include <string>

int main(int argc, char* argv[])
{
	string path = "D:\\Master\\ImageProcessing\\Data\\LenaPeppersBaboon\\";
	string fname = "LenaPeppersBaboon.txt";
	int M = 2;
	int max_iter = 300;
	double epsilon = 1e-10;//numeric_limits<double>::epsilon();
	int mode = 1;
	int stop_iter = INT_MAX;
	FILE * fp;

	if (argc > 2){
		path = string(argv[1]);
		fname = string(argv[2]);

		if (argc > 3){
			M = atoi(argv[3]);

			if (argc > 4){
				max_iter = atoi(argv[4]);

				if (argc > 5){
					epsilon = atof(argv[5]);

					if (argc > 6){
						mode = atoi(argv[6]);

						if (argc > 7){
							stop_iter = atoi(argv[7]) - 1;
						}
						else if (argc > 8){
							fp = fopen(argv[8], "a");
						}
					}
				}
			}
		}
	}
	time_t rawtime;
	time ( &rawtime );
	GFKM G(path, fname);
	G.epsilon = epsilon;
	G.max_iter = max_iter;
	G.M = M;
	G.initialize_centroids();

	if(argc < 9) 
		fp = fopen("GFKM7.test.log", "a");
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# %s", ctime(&rawtime));
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp," %s\%s\n N = %d, D = %d, K = %d, M = %d, epsilon = %.0e\n Mode = %s\n", 
		path.c_str(), fname.c_str(), G.N, G.D, G.K, G.M, G.epsilon, 
		mode==0 ? "Updating centroids by CPU" : "Updating centroids by GPU");
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# GPU running\n");
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	double * gpu_rs = GFKM_GPU(fp, G, 256, stop_iter, mode);
	G.J = 0.0;
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# CPU running\n");
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	double * cpu_rs = G.run(fp, (int)gpu_rs[1]);
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# Total CPU time      : %9.2lf\n", cpu_rs[0]);
	fprintf(fp, "# Total GPU time      : %9.2lf\n", gpu_rs[0]);
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# Speedup             : %9.2lf\n", cpu_rs[0]/gpu_rs[0]);
	fclose(fp);
	printf(" Speedup: %.2lf\n", cpu_rs[0]/gpu_rs[0]);
	delete [] gpu_rs;
	delete [] cpu_rs;
	
	//getch();
    return 0;
}