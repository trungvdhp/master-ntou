#include "hip/hip_runtime.h"
#include "SFKM.cuh"
#include "FKM.h"
#include "TimingCPU.h"
#include <conio.h>

int main(int argc, char* argv[])
{
	string path = "D:\\Master\\ImageProcessing\\Data\\LenaPeppersBaboon\\";
	string fname = "LenaPeppersBaboon.txt";
	int max_iter = 300;
	int stop_iter = INT_MAX;
	int mode = 1;
	double epsilon = 1e-8;//numeric_limits<double>::epsilon();
	FILE * fp;

	if (argc > 2){
		path = string(argv[1]);
		fname = string(argv[2]);

		if (argc > 3){
			max_iter = atoi(argv[3]);

			if (argc > 4){
				epsilon = atof(argv[4]);

				if (argc > 5){
					mode = atoi(argv[5]);

					if (argc > 6){
						stop_iter = atoi(argv[6]) - 1;

						if (argc > 7){
							fp = fopen(argv[7], "a");
						}
					}
				}
			}
		}
	}
	time_t rawtime;
	time ( &rawtime );
	FKM G(path, fname);
	G.epsilon = epsilon;
	G.max_iter = max_iter;
	G.initialize_centroids();

	if(argc < 8) 
		fp = fopen("SFKM.test.log", "a");
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# %s", ctime(&rawtime));
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp," %s%s\n N = %d, D = %d, K = %d, epsilon = %.0e\n", 
		path.c_str(), fname.c_str(), G.N, G.D, G.K, G.epsilon);
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# GPU running\n");
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	double * gpu_rs = FKM_GPU(fp, G, 256, stop_iter, mode);
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# CPU running\n");
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	double * cpu_rs = G.run(fp, (int)gpu_rs[1]);
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# Total CPU time      : %9.2lf\n", cpu_rs[0]);
	fprintf(fp, "# Total GPU time      : %9.2lf\n", gpu_rs[0]);
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# Speedup             : %9.2lf\n", cpu_rs[0]/gpu_rs[0]);
	fclose(fp);
	printf(" Speedup: %.2lf\n", cpu_rs[0]/gpu_rs[0]);
	delete [] gpu_rs;
	delete [] cpu_rs;
	
	//getch();
    return 0;
}