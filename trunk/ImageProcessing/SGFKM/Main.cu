#include "hip/hip_runtime.h"
#include "SGFKM.cuh"
#include "GFKM.h"
#include "TimingCPU.h"
#include <conio.h>
#include <chrono>

int main(int argc, char* argv[])
{
	std::string path = "D:\\Master\\ImageProcessing\\Data\\LenaPeppersBaboon\\"; //LenaPeppersBaboon
	std::string fname = "LenaPeppersBaboon.txt";
	int M = 2;
	int max_iter = 1;
	int stop_iter = 0;
	int test_runs = 1;
	int mode = 2;
	int block_size = 1;
	/*printf("Input block size : ");
	scanf("%d", &block_size);
	printf("Input mode : ");
	scanf("%d", &mode);
	printf("Input stop iter : ");
	scanf("%d", &stop_iter);*/
	//printf("Input file name : ");
	//scanf("%s", &fname);
	double epsilon = 1e-8;//numeric_limits<double>::epsilon();
	FILE * fp;

	if (argc > 2){
		path = std::string(argv[1]);
		fname = std::string(argv[2]);

		if (argc > 3){
			M = atoi(argv[3]);

			if (argc > 4){
				max_iter = atoi(argv[4]);

				if (argc > 5){
					epsilon = atof(argv[5]);

					if (argc > 6){
						mode = atoi(argv[6]);

						if (argc > 7){
							stop_iter = atoi(argv[7]) - 1;

							if (argc > 8){
								block_size = atoi(argv[8]);

								if (argc > 9){
									test_runs = atoi(argv[9]);

									if (argc > 10){
										fp = fopen(argv[10], "a");
									}
								}
							}
						}
					}
				}
			}
		}
	}
	time_t rawtime;
	time ( &rawtime );
	GFKM G(path, fname, M);
	G.epsilon = epsilon;
	G.max_iter = max_iter;

	if(argc < 11) 
		fp = fopen("SGFKM.test.log", "a");
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp, "# %s", ctime(&rawtime));
	fprintf(fp, "-------------------------------------------------------------------------------\n");
	fprintf(fp," %s%s\n N = %d, D = %d, K = %d, M = 2,  epsilon = %.0e, block size = %d, test runs = %d.\n", 
		path.c_str(), fname.c_str(), G.N, G.D, G.K, G.epsilon, block_size, test_runs);
	//fprintf(fp," %s%s\n, M = 2, block size = %d, test runs = %d.\n", 
		//path.c_str(), fname.c_str(), block_size, test_runs);

	int sleeping_time = 5000;
	int warpSize = 32;
	int block_sizes[32];
	for (int i = 0, j = 32; i < 32; ++i, j += warpSize) block_sizes[i] = j;
	int start_id = 17;
	int end_id = 32;

	if (block_size <= 0 || block_size >= 32 && block_size <= 1024)
	{
		start_id = 0;
		end_id = 1;
		block_sizes[0] = block_size;
	}
	G.initialize_centroids();

	for (; start_id < end_id; start_id++)
	{
		fprintf(fp, "-------------------------------------------------------------------------------\n");
		fprintf(fp, "# Run with block size = %d\n", block_sizes[start_id]);
		printf ("# Run with block size = %d\n", block_sizes[start_id]);

		int start_mode = 1, end_mode = 5;
	
		if (mode > 0 && mode < 5){
			start_mode = mode;
			end_mode = start_mode + 1;
		}
		else
		{
			fprintf(fp, "-------------------------------------------------------------------------------\n");
			fprintf(fp, "# Mode %d is not from 1 to 4, so run all modes:\n", mode);
			printf ("# Mode %d is not from 1 to 4, so run all modes:\n", mode);
		}
		double * cpu_rs = G.run(fp, stop_iter);
		G.restore_initial_centroids();

		while (start_mode < end_mode)
		{
			fprintf(fp, "-------------------------------------------------------------------------------\n");
			fprintf(fp, "# Mode %d :\n", start_mode);
			printf ("# Running mode %d :\n", start_mode);
			double total_speedup = 0.0, step1_speedup = 0.0, step2_speedup = 0.0, step3_speedup = 0.0;
			printf ("# Test run = %d\n",test_runs);

			for (int i = 0; i < test_runs; ++i)
			{
				
				//printf("Finished CPU running #%d!\n", i+1);
				//_sleep(sleeping_time);
				//printf("Stop iteration = %d\n", (int)cpu_rs[3]);
				double * gpu_rs = GFKM_GPU(fp, G, block_sizes[start_id], (int)cpu_rs[3], start_mode);

				total_speedup = total_speedup + (cpu_rs[0] + cpu_rs[1] + cpu_rs[2]) / (gpu_rs[0] + gpu_rs[1] + gpu_rs[2] + gpu_rs[3]);
				step1_speedup = step1_speedup + cpu_rs[0] / gpu_rs[0];
				step2_speedup = step2_speedup + cpu_rs[1] / gpu_rs[1];
				step3_speedup = step3_speedup + cpu_rs[2] / gpu_rs[2];
				
				delete [] gpu_rs;
			}
			fprintf(fp, "-------------------------------------------------------------------------------\n");
			fprintf(fp, "# Step1 average speedup of %d times: %9.2lf\n", test_runs, step1_speedup/test_runs);
			fprintf(fp, "# Step2 average speedup of %d times: %9.2lf\n", test_runs, step2_speedup/test_runs);
			fprintf(fp, "# Step3 average speedup of %d times: %9.2lf\n", test_runs, step3_speedup/test_runs);
			fprintf(fp, "# Total average speedup of %d times: %9.2lf\n", test_runs, total_speedup/test_runs);
			printf("# Step1 average speedup of %d times: %9.2lf\n", test_runs, step1_speedup/test_runs);
			printf("# Step2 average speedup of %d times: %9.2lf\n", test_runs, step2_speedup/test_runs);
			printf("# Step3 average speedup of %d times: %9.2lf\n", test_runs, step3_speedup/test_runs);
			printf("# Total average speedup of %d times: %9.2lf\n", test_runs, total_speedup/test_runs);

			//printf(" Sleeping in %d seconds...\n", sleeping_time / 1000);
			_sleep(sleeping_time);
			start_mode++;
		}
		delete [] cpu_rs;
	}
	
	fclose(fp);
	//getch();
    return 0;
}