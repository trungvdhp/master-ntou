#include "hip/hip_runtime.h"
#include "SGFKM.cuh"
#include "Util.h"
#define DIM_MAX 16
#define MMAX 32
#define NSTREAM 5

inline __host__ int roundup(int x, int y)
{
	return 1 + (x-1)/y;
}

__global__ void update_memberships_kernel(
	double * points, double * centroids, double * memberships, int * NNT,
	int N, int D, int K, int M, double fuzzifier)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= N) return;
	int i, j;
	int * pNNT = NNT + idx*M;

	double * pMemberships = memberships + idx*K;
	double * pCentroids = centroids;
	
	double X[DIM_MAX];
	double DNNT[MMAX];

	double f = 1. / (fuzzifier - 1.);
	double diff, temp, sum = 0.;

	for (i = 0, j = idx*D; i < D; ++i, ++j) X[i] = points[j];

	for (i = 0; i < M; ++i) DNNT[i] = DBL_MAX;

	for (i = 0; i < K; ++i, pCentroids += D){
		diff = 0.;

		for (j = 0; j < D; ++j){
			temp = X[j] - pCentroids[j];
			diff = diff + temp*temp;
		}
		idx = 0;

		for (; idx < M; ++idx){
			if (DNNT[idx] > diff) break;
		}

		for (j = M-1; j > idx; --j){
			DNNT[j] = DNNT[j-1];
			pNNT[j] = pNNT[j-1];
		}

		if (idx < M){
			DNNT[idx] = diff;
			pNNT[idx] = i;
		}
	}
	for (i = 0; i < K; ++i) pMemberships[i] = 0.;

	for (i = 0; i < M; ++i){
		diff = DNNT[i];

		if (diff == 0.){ 
			pMemberships[pNNT[i]] = 1.;
			return;
		}
		diff = pow(diff, f);
		pMemberships[pNNT[i]] = diff;
		sum = sum + 1. / diff;
	}

	for (i = 0; i < M; ++i){
		pMemberships[pNNT[i]] = pow(pMemberships[pNNT[i]]*sum, -fuzzifier);
	}
}

__global__ void reduce_memberships_kernel(double * memberships, double * odata, int N)
{
	extern __shared__ double sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x*blockDim.x + tid;
	int gridSize = blockDim.x*gridDim.x;
	double temp = 0.0;

	while(i < N){
		temp = temp + memberships[i];
		i += gridSize;
	}
	sdata[tid] = temp;
	__syncthreads();

	//if (blockDim.x > 511){
	//	if (tid < 256)
	//		sdata[tid] = sdata[tid] + sdata[tid+256];
	//	__syncthreads();
	//}

	if (blockDim.x > 255){
		if (tid < 128) sdata[tid] = sdata[tid] + sdata[tid+128];
		__syncthreads();
	}

	if (blockDim.x > 127){
		if (tid < 64) sdata[tid] = sdata[tid] + sdata[tid+64];
		__syncthreads();
	}

	if (tid < 32) sdata[tid] = sdata[tid] + sdata[tid + 32];
	if (tid < 16) sdata[tid] = sdata[tid] + sdata[tid + 16];
	if (tid < 8) sdata[tid] = sdata[tid] + sdata[tid + 8];
	if (tid < 4) sdata[tid] = sdata[tid] + sdata[tid + 4];
	if (tid < 2) sdata[tid] = sdata[tid] + sdata[tid + 2];

	if (tid == 0) odata[blockIdx.x] = sdata[0] + sdata[1];
}

__global__ void reduce_centroids_kernel
	(double * points, double * memberships, double * odata, int N)
{
	extern __shared__ double sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x*blockDim.x + tid;
	int gridSize = blockDim.x*gridDim.x;
	double temp = 0.0;
	
	while(i < N){
		temp = temp + points[i] * memberships[i];
		i += gridSize;
	}
	sdata[tid] = temp;
	__syncthreads();

	if (tid < 128) 
		sdata[tid] = sdata[tid] + sdata[tid+128];
	__syncthreads();

	if (tid < 64) 
		sdata[tid] = sdata[tid] + sdata[tid+64];
	__syncthreads();

	if (tid < 32) sdata[tid] = sdata[tid] + sdata[tid + 32];
	if (tid < 16) sdata[tid] = sdata[tid] + sdata[tid + 16];
	if (tid < 8) sdata[tid] = sdata[tid] + sdata[tid + 8];
	if (tid < 4) sdata[tid] = sdata[tid] + sdata[tid + 4];
	if (tid < 2) sdata[tid] = sdata[tid] + sdata[tid + 2];

	if (tid == 0) odata[blockIdx.x] = sdata[0] + sdata[1];
}

__global__ void calculate_new_centroids(double * centroids, double * memberships)
{
	int cid = blockIdx.x*blockDim.x + threadIdx.x;
	centroids[cid] = centroids[cid] / memberships[blockIdx.x];
}

__host__ void calculate_new_centroids(
	double * points, double * memberships, double * newCentroids, 
	int * NNT, int N, int D, int K, int M)
{
	int i, j, k, idx;
	int * pNNT = NNT;
	double * pMemberships = memberships;
	double * pPoints = points;
	double * pCentroids;
	double * sum = new double[K]();
	memset(newCentroids, 0, K*D*sizeof(double));

	for (i = 0; i < N; ++i, pMemberships += K, pNNT += M, pPoints += D){
		for (j = 0; j < M; ++j){
			idx = pNNT[j];
			sum[idx] = sum[idx] + pMemberships[idx];
			pCentroids = newCentroids + idx*D;

			for (k=0; k<D; ++k)
				pCentroids[k] = pCentroids[k] + pMemberships[idx]*pPoints[k];
		}
	}
	pCentroids = newCentroids;

	for (i = 0; i < K; ++i, pCentroids += D)
		for (j = 0; j < D; ++j)
			pCentroids[j] = pCentroids[j] / sum[i];
}

__global__ void check_convergence(double * centroids, double * newCentroids, int * flag, double epsilon)
{
	flag[0] = 0;

	for (int i = 0; i < blockDim.x; ++i){
		if (fabs(centroids[i] - newCentroids[i]) >= epsilon) return;
	}
	flag[0] = 1;
}

__host__ double * FKM_GPU(FILE * f, GFKM & G, int block_size, int stop_iter, int mode)
{
#pragma region Declare common variables
	int i, j, k, x, y, z, w;
	int DBL_SIZE = sizeof(double);
	
	int sizeC = G.K * G.D;
	int points_size = G.N * G.D * DBL_SIZE;
	int c_size = G.K * DBL_SIZE;
	int centroids_size = c_size * G.D;
	int uk_size = G.N * c_size;
	int NNT_size = G.N * G.M * DBL_SIZE;
	int sm_size = block_size * DBL_SIZE;
	int block_dsize = block_size<<2;
	int num_blocks = roundup(G.N, block_size);
	int num_cblocks = roundup(G.N, block_dsize);
	int u_size = num_cblocks * c_size;
	int tempC_size = num_cblocks * centroids_size;
	double t1 = 0.0, t2 = 0.0, t3 = 0.0, t4;
	
	TimingCPU tmr_CPU;
	TimingGPU tmr_GPU;

	double alpha, beta;
	double * p1;
	double * p2;
	double * p3;
#pragma endregion

#pragma region Declare device memories
	double * d_points;
	double * d_pointsT;
	double * d_centroids;
	double * d_memberships;
	double * d_membershipsT;
	double * d_u;
	double * d_sumU;
	double * d_tempC;
	int * d_stop;
	int * d_NNT;
#pragma endregion

#pragma region Declare host pinned memories
	double * p_points;
	double * p_centroids;
	double * p_memberships;
	double * p_u;
	double * p_sumU;
	double * p_tempC;
	int * p_stop;
	int * p_NNT;
#pragma endregion

#pragma region Malloc device
	CudaSafeCall(hipMalloc(&d_points, points_size));
	CudaSafeCall(hipMalloc(&d_pointsT, points_size));
	CudaSafeCall(hipMalloc(&d_centroids, centroids_size));
	CudaSafeCall(hipMalloc(&d_memberships, uk_size));
	CudaSafeCall(hipMalloc(&d_membershipsT, uk_size));
	CudaSafeCall(hipMalloc(&d_u, u_size));
	CudaSafeCall(hipMalloc(&d_sumU, c_size));
	CudaSafeCall(hipMalloc(&d_tempC, tempC_size));
	CudaSafeCall(hipMalloc(&d_stop, sizeof(int)));
	CudaSafeCall(hipMalloc(&d_NNT, NNT_size));
#pragma endregion

#pragma region Malloc host
	CudaSafeCall(hipHostMalloc(&p_points, points_size));
	CudaSafeCall(hipHostMalloc(&p_centroids, centroids_size));
	CudaSafeCall(hipHostMalloc(&p_memberships, uk_size));
	CudaSafeCall(hipHostMalloc(&p_u, u_size));
	CudaSafeCall(hipHostMalloc(&p_sumU, c_size));
	CudaSafeCall(hipHostMalloc(&p_tempC, tempC_size));
	CudaSafeCall(hipHostMalloc(&p_stop, sizeof(int)));
	CudaSafeCall(hipHostMalloc(&p_NNT, NNT_size));
#pragma endregion

#pragma region Memories copy
	memcpy(p_points, G.points, points_size);
	memcpy(p_centroids, G.centroids, centroids_size);
	CudaSafeCall(hipMemcpy(d_points, p_points, points_size, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_centroids, p_centroids, centroids_size, hipMemcpyHostToDevice));  
#pragma endregion

#pragma region Declare cuda streams and transpose points
	hipblasHandle_t handle;
	hipStream_t * streams = new hipStream_t[NSTREAM];

	if (mode == 1){
		for (i = 0; i < NSTREAM; ++i)
			hipStreamCreate(&streams[i]);

		CublasSafeCall(hipblasCreate(&handle));
		alpha = 1.;
		beta  = 0.;
		tmr_GPU.StartCounter();
		CublasSafeCall(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, G.N, G.D,
			&alpha, d_points, G.D, &beta, d_points, G.D, d_pointsT, G.N)); 
		t2 = t2 + tmr_GPU.GetCounter();
	}
#pragma endregion

#pragma region Main loop
	for (i = 0; i< G.max_iter; ++i){
#pragma region  Update memberships by GPU
		tmr_GPU.StartCounter();
		update_memberships_kernel<<<num_blocks, block_size>>>
			(d_points, d_centroids,d_memberships, d_NNT, G.N, G.D, G.K, G.M, G.fuzzifier);
		//CudaCheckError();
		t1 = t1 + tmr_GPU.GetCounter();
#pragma endregion
		
		if (mode == 1){
#pragma region Transpose memberships
			alpha = 1.;
			beta  = 0.;
			tmr_GPU.StartCounter();
			CublasSafeCall(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, G.N, G.K,
				&alpha, d_memberships, G.K, &beta, d_memberships, G.K, d_membershipsT, G.N)); 
			t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

#pragma region Reduce centroids by GPU
			tmr_GPU.StartCounter();
			for (j = 0, x = 0, y = 0, z = 0; j < G.K; ++j, x += G.N, y += num_cblocks){
				reduce_memberships_kernel<<<num_cblocks, block_size, sm_size, streams[0]>>>
					(d_membershipsT + x, d_u + y, G.N);

				for (k = 0, w = 0; k < G.D; ++k, w += G.N, z += num_cblocks){
					reduce_centroids_kernel<<<num_cblocks, block_size, sm_size, streams[k % (NSTREAM-1)+1]>>>
							(d_pointsT + w, d_membershipsT + x, d_tempC + z, G.N);
				}
			}
			t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion

#pragma region Reduce block sums by CPU
			if (num_cblocks > 1){
				tmr_GPU.StartCounter();
				CudaSafeCall(hipMemcpyAsync(p_u, d_u, u_size, hipMemcpyDeviceToHost));
				CudaSafeCall(hipMemcpyAsync(p_tempC, d_tempC, tempC_size, hipMemcpyDeviceToHost));
				t2 = t2 + tmr_GPU.GetCounter();
				tmr_CPU.start();

				for (j = 0, p1 = p_u, p2 = p_tempC, p3 = p_centroids; j < G.K; ++j, p1 += num_cblocks, p3 += G.D){
					p_sumU[j] = 0.0;

					for (x = 0; x < num_cblocks; ++x)
						p_sumU[j] = p_sumU[j] + p1[x];

					for (x = 0; x < G.D; ++x, p2 += num_cblocks){
						p3[x] = 0.0;

						for (y = 0; y < num_cblocks; ++y){
							p3[x] = p3[x] + p2[y];
						}
					}
				}
				tmr_CPU.stop();
				t2 = t2 + tmr_CPU.elapsed();
				tmr_GPU.StartCounter();
				CudaSafeCall(hipMemcpyAsync(d_u, p_sumU, c_size, hipMemcpyHostToDevice));
				CudaSafeCall(hipMemcpyAsync(d_tempC, p_centroids, centroids_size, hipMemcpyHostToDevice));
				t2 = t2 + tmr_GPU.GetCounter();
			}
#pragma endregion

#pragma region Calculate centroids by GPU
			tmr_GPU.StartCounter();
			calculate_new_centroids<<<G.K, G.D>>>(d_tempC, d_u);
			t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion
		}
		else{
#pragma region Calculate centroids by CPU
			tmr_GPU.StartCounter();
			CudaSafeCall(hipMemcpyAsync(p_NNT, d_NNT, NNT_size, hipMemcpyDeviceToHost));
			CudaSafeCall(hipMemcpyAsync(p_memberships, d_memberships, uk_size, hipMemcpyDeviceToHost));
			t2 = t2 + tmr_GPU.GetCounter();
			tmr_CPU.start();
			calculate_new_centroids(p_points, p_memberships, p_centroids, p_NNT, G.N, G.D, G.K, G.M);
			tmr_CPU.stop();
			t2 = t2 + tmr_CPU.elapsed();
			tmr_GPU.StartCounter();
			CudaSafeCall(hipMemcpyAsync(d_tempC, p_centroids, centroids_size, hipMemcpyHostToDevice));
			t2 = t2 + tmr_GPU.GetCounter();
#pragma endregion
		}
#pragma region Getting and checking stop-condition
		tmr_GPU.StartCounter();
		check_convergence<<<1, sizeC>>>(d_centroids, d_tempC, d_stop, G.epsilon);
		CudaSafeCall(hipMemcpyAsync(p_stop, d_stop, sizeof(int), hipMemcpyDeviceToHost));
		t3 = t3 + tmr_GPU.GetCounter();
		tmr_GPU.StartCounter();
		CudaSafeCall(hipMemcpyAsync(d_centroids, d_tempC, centroids_size, hipMemcpyDeviceToDevice));
		t2 = t2 + tmr_GPU.GetCounter();

		if ((p_stop[0] == 1 && (stop_iter == INT_MAX || i==stop_iter)) || i==stop_iter)
			break;
#pragma endregion
	}
	if (i == G.max_iter) i--;
#pragma endregion

#pragma region Copying device back to host
	tmr_GPU.StartCounter();
	CudaSafeCall(hipMemcpyAsync(p_centroids, d_centroids, centroids_size, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpyAsync(p_NNT, d_NNT, NNT_size, hipMemcpyDeviceToHost));
	t4 = tmr_GPU.GetCounter();
#pragma endregion

#pragma region Writing results to files
	Util::write<double>(p_centroids, G.K, G.D, G.path + "centroids.GPU.txt");
	Util::write<int>(p_NNT, G.N, G.M, G.path + "NNT.GPU.txt");
	Util::print_times(f, t1, t2, t3, i+1);
#pragma endregion

#pragma region Cuda free device memories
	hipFree(d_points);
	hipFree(d_pointsT);
	hipFree(d_centroids);
	hipFree(d_memberships);
	hipFree(d_membershipsT);
	hipFree(d_u);
	hipFree(d_sumU);
	hipFree(d_tempC);
	hipFree(d_stop);
	hipFree(d_NNT);
#pragma endregion

#pragma region Cuda free host pinned memories
	hipHostFree(p_points);
	hipHostFree(p_centroids);
	hipHostFree(p_memberships);
	hipHostFree(p_u);
	hipHostFree(p_sumU);
	hipHostFree(p_tempC);
	hipHostFree(p_stop);
	hipHostFree(p_NNT);
#pragma endregion

#pragma region Get total time and last iteration index
	double * rs = new double[2];
	rs[0] = t1 + t2 + t3 + t4;
	rs[1] = (double)i;
#pragma endregion

#pragma region CublasDestroy, CudaStreamDestroy, and DeviceReset
	if (mode == 1)
	{
		CublasSafeCall(hipblasDestroy(handle));

		for (i = 0; i < NSTREAM; ++i)
			hipStreamDestroy(streams[i]);
	}
	
	hipDeviceReset();
#pragma endregion
	
	return rs;
}